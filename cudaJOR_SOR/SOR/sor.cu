#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define GAMMA 0.5

double r2();
__global__ void JORkernel(double *cudaM,double *cudaX, int dim)
{
        //int idx = blockIdx.x * blockDim.x + threadIdx.x;        // Finds the thread_id
        int blocksize=16;
        int gridsize=gridDim.x; //this will always be 1
       // int gridsize2=gridDim.x;
       // printf("gridsize: %d %d\n", gridsize,gridsize2);
       // __device__ bool myGlobalFlag=true;
       
        int bx=blockIdx.x;
        int by=blockIdx.y;
        int tx=threadIdx.x;
        int ty=threadIdx.y;
        
        int blocktotal=blocksize*blocksize;
        int temp1=by*gridsize+bx;
        int temp2=ty*blocksize+tx;
        int j;
        int i;

       int ind=temp1*blocktotal+temp2;
       if(ind<dim)
     { 
      //put JOR formula here
     	//b is 0 all the time so we don't need to consider about it so far
     	int tempi=ind%dim;
     	double sum=0;
      //this is the difference between SOR and JOR:
      double partialsum=0;
      double partialsum1=0;
      double* temparray;
      temparray= (double*)malloc((tempi-1)*sizeof(double));
      //this will be calcauted serializely
      //temparray[0] correspond to the case the i=0
        for(j=1;j<dim;j++)
        {

           partialsum=partialsum+cudaM[0+j]*cudaX[j];
           
        }
         //sicne we only update after the sync so the cudaX[0] here is still the old value.
      temparray[0]=(1-GAMMA)*cudaX[0]-GAMMA/cudaM[0]*partialsum;
      //then use temparray to serializely getting others!
      for(j=1;j<=tempi-1;j++)
      {
        partialsum=0;
        partialsum1=0;
        for(i=0;i<j;i++)
        {
          partialsum=partialsum+cudaM[j*dim+i]*temparray[i];
        }
        for(i=j+1;i<dim;i++)
        {
          partialsum1=partialsum1+cudaM[j*dim+i]*cudaX[i];
        }
        temparray[j]=(1-GAMMA)*cudaX[j]-GAMMA/cudaM[j*dim+j]*(partialsum+partialsum1);
      }

     	for(j=tempi+1;j<dim;j++)
     	{
     	 sum=sum+cudaM[tempi*dim+j]*cudaX[j];
     	}
      for(j=0;j<=tempi-1;j++)
      {
        sum=sum+cudaM[tempi*dim+j]*temparray[j];
      }
    __syncthreads();
     cudaX[ind]=(1-GAMMA)*cudaX[ind]-GAMMA/cudaM[tempi*dim+tempi]*sum; //temp is the updated x, do the update in order to make sure the serialized step use the old value in this way.
     }
     __syncthreads();
     //wait for all the threads to finish, this is not going to work because it only snyc threads inside of one block.
}

int main(int argc, char *argv[])
{
   
   if( argc == 2 ) {
      printf("The matrix dimension is %s\n", argv[1]);
   }
   else if( argc > 2 ) {
      printf("Too many arguments supplied.\n");
   }
   else {
      printf("One argument expected.\n");
   }

        int i;
        int j;
        int dim=atoi(argv[1]);
        double *matrix;
        double *x;
        double *previousx; // to install the previous x for compare purpose
        int loopCount=0; //use to see how many iteration we need to get the correct result;
        double tolerance=0.01;
       /* double *b;*/
        matrix=(double*) malloc(dim*dim*sizeof(double));
      /*  b=(double*) malloc(dim*sizeof(double));*/
        x=(double*) malloc(dim*sizeof(double));
        previousx=(double*) malloc(dim*sizeof(double));
        //the diagonal dominated matrix will be automatically be not singular!!
      for(i=0;i<dim;i++)
      {
        double rowSum=0.0;
        for(j=0;j<dim;j++)
        {
            
            matrix[i*dim+j]=r2();
            rowSum=matrix[i*dim+j]+rowSum;
        }
        matrix[i*dim+i]=rowSum;
      }
      //x will all be initilized to be 1
      for(i=0;i<dim;i++)
      {
      	x[i]=1.0;
      	previousx[i]=1.0;
      	//b[i]=0.0;  //make b to be 0 as initial then easy to check that the value x should goes to 0;
      }
      
  //print the matrix out to check
     // for(i=0;i<dim;i++)
     // {
       // for(j=0;j<dim;j++)
      //  {
        //  printf("%lf ",matrix[i*dim+j]);
       // }
       // printf("\n");
     // }
      double *cudaM; //prepare for cuda global memory
      double *cudaX;
   /*   double *cudaB;*/
      int xsize=dim*sizeof(double);
      int msize=dim*dim*sizeof(double);
      hipMalloc((void**)&cudaM,msize);
      hipMalloc((void**)&cudaX,xsize);
    /*  cudaMalloc((void**)&cudaB,xsize);*/ //b have the same size with x



     //start timing here
      clock_t begin,end;
        begin=clock();
      hipMemcpy(cudaM,matrix,msize,hipMemcpyHostToDevice);
      hipMemcpy(cudaX,x,xsize,hipMemcpyHostToDevice);

      int blocksize=16;
      int gridsize= dim/256+1; //make the gridsize to be an ro  like (1,2)
      dim3 Grid( 1, gridsize);                   // Number of threads per block
      dim3 Block( blocksize,blocksize);              // Number of thread blocks

      bool stopFlag=false;
      while (!stopFlag)
      {
      loopCount=loopCount+1;
      JORkernel<<<Grid, Block>>>(cudaM,cudaX,dim);
      hipMemcpy( x, cudaX, xsize, hipMemcpyDeviceToHost);
      //comapre new x with previous x
      for(i=0;i<dim;i++)
      {
      	if((x[i]-previousx[i]>=tolerance)||(previousx[i]-x[i])>=tolerance) //check if the current value and previous value is close enough
      		//if not we need to keep going
      	{
      		for(j=0;j<dim;j++)
      		{
      			previousx[j]=x[j];
      		}
      		stopFlag=false;

      		break;
      	}
        if(i==dim-1) //this means there is no break in the middle at all
       {
      	stopFlag=true;
       }

      } 
     }
       double time_spent;
        end=clock();
       time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
     printf("matrix size: %d-iteration times: %d-error tolerance set to:%lf \n",dim,loopCount,tolerance);
     printf("time spent:%lf seconds \n",time_spent);  
    for(i=0;i<dim;i++)
      {
       if(i%10==0)
      {
       printf("\n");
       }
       printf("[%d]:%lf ",i,x[i]);
      }
      hipFree(cudaX); 
      hipFree(cudaM);

     return 0;
}

double r2()
{
    return (double)rand() / (double)RAND_MAX ;
}







