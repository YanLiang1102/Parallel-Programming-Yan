#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define GAMMA 0.5

double r2();
__global__ void JORkernel(double *cudaM,double *cudaX, int dim)
{
        //int idx = blockIdx.x * blockDim.x + threadIdx.x;        // Finds the thread_id
        int blocksize=16;
        int gridsize=gridDim.x; //this will always be 1
       // int gridsize2=gridDim.x;
       // printf("gridsize: %d %d\n", gridsize,gridsize2);
       // __device__ bool myGlobalFlag=true;
       
        int bx=blockIdx.x;
        int by=blockIdx.y;
        int tx=threadIdx.x;
        int ty=threadIdx.y;
        
        int blocktotal=blocksize*blocksize;
        int temp1=by*gridsize+bx;
        int temp2=ty*blocksize+tx;
        int j;
        //bool localFlag; // idea is to "and" all the flag and see any false 
       // printf("bx: %d \n",bx);
       // printf("by: %d \n",by);
       // printf("tx: %d \n",tx);
       // printf("ty: %d \n",ty);

        //double temp; //this temp is going to check with the old x and see if closer enough if not it will be copied into cudaX;

       int ind=temp1*blocktotal+temp2;
       if(ind<dim)
     { 
       //printf("%d \n",ind);
      //put JOR formula here
     	//b is 0 all the time so we don't need to consider about it so far
     	int tempi=ind%dim;
     	double sum=0;
     	for(j=0;j<dim;j++)
     	{
     		if(j!=tempi)
     		{
     			sum=sum+cudaM[tempi*dim+j]*cudaX[j];
     		}
     	}
     	cudaX[ind]=(1-GAMMA)*cudaX[ind]-GAMMA/cudaM[tempi*dim+tempi]*sum; //temp is the updated x
        //now comapre
/*        if(temp-cudaX[ind]>=0.01||cudaX[ind]-temp>=0.01)
        {
        	localFlag=false;
        }
        else
        {
        	localFlag=true;
        }
        myGlobalFlag=myGlobalFlag && localFlag;*/
     }

     __syncthreads();
     //wait for all the threads to finish, this is not going to work because it only snyc threads inside of one block.
}

int main(int argc, char *argv[])
{
   
   if( argc == 2 ) {
      printf("The matrix dimension is %s\n", argv[1]);
   }
   else if( argc > 2 ) {
      printf("Too many arguments supplied.\n");
   }
   else {
      printf("One argument expected.\n");
   }

        int i;
        int j;
        int dim=atoi(argv[1]);
        double *matrix;
        double *x;
        double *previousx; // to install the previous x for compare purpose
        int loopCount=0; //use to see how many iteration we need to get the correct result;
        double tolerance=0.001;
       /* double *b;*/
        matrix=(double*) malloc(dim*dim*sizeof(double));
      /*  b=(double*) malloc(dim*sizeof(double));*/
        x=(double*) malloc(dim*sizeof(double));
        previousx=(double*) malloc(dim*sizeof(double));
        //the diagonal dominated matrix will be automatically be not singular!!
      for(i=0;i<dim;i++)
      {
        double rowSum=0.0;
        for(j=0;j<dim;j++)
        {
            
            matrix[i*dim+j]=r2();
            rowSum=matrix[i*dim+j]+rowSum;
        }
        matrix[i*dim+i]=rowSum;
      }
      //x will all be initilized to be 1
      for(i=0;i<dim;i++)
      {
      	x[i]=1.0;
      	previousx[i]=1.0;
      	//b[i]=0.0;  //make b to be 0 as initial then easy to check that the value x should goes to 0;
      }
      
  //print the matrix out to check
     // for(i=0;i<dim;i++)
     // {
       // for(j=0;j<dim;j++)
      //  {
        //  printf("%lf ",matrix[i*dim+j]);
       // }
       // printf("\n");
     // }
      double *cudaM; //prepare for cuda global memory
      double *cudaX;
   /*   double *cudaB;*/
      int xsize=dim*sizeof(double);
      int msize=dim*dim*sizeof(double);
      hipMalloc((void**)&cudaM,msize);
      hipMalloc((void**)&cudaX,xsize);
    /*  cudaMalloc((void**)&cudaB,xsize);*/ //b have the same size with x



     //start timing here
      clock_t begin,end;
        begin=clock();
      hipMemcpy(cudaM,matrix,msize,hipMemcpyHostToDevice);
      hipMemcpy(cudaX,x,xsize,hipMemcpyHostToDevice);
  /*    cudaMemcpy(cudaB,b,xsize,cudaMemcpyHostToDevice);*/

     // cudaMalloc( (void**)&ad, csize );
     // cudaMalloc( (void**)&bd, isize );
     // cudaMemcpy( ad, a, csize, cudaMemcpyHostToDevice );
     // cudaMemcpy( bd, b, isize, cudaMemcpyHostToDevice );
      int blocksize=16;
      int gridsize= dim/256+1; //make the gridsize to be an ro  like (1,2)
      dim3 Grid( 1, gridsize);                   // Number of threads per block
      dim3 Block( blocksize,blocksize);              // Number of thread blocks

      bool stopFlag=false;
      while (!stopFlag)
      {
      loopCount=loopCount+1;
      JORkernel<<<Grid, Block>>>(cudaM,cudaX,dim);
      hipMemcpy( x, cudaX, xsize, hipMemcpyDeviceToHost);
      //comapre new x with previous x
      for(i=0;i<dim;i++)
      {
      	if((x[i]-previousx[i]>=tolerance)||(previousx[i]-x[i])>=tolerance) //check if the current value and previous value is close enough
      		//if not we need to keep going
      	{
      		for(j=0;j<dim;j++)
      		{
      			previousx[j]=x[j];
      		}
      		stopFlag=false;
      		//don't need to copy to cuda again since it will be the same cudaX.
      		//JORKernel<<<Grid,Block>>>(cudaM,cudaX,dim);
      		break;
      	}
        if(i==dim-1) //this means there is no break in the middle at all
       {
      	stopFlag=true;
       }

      } 
     }
       double time_spent;
        end=clock();
       time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
     printf("matrix size: %d-iteration times: %d-error tolerance set to:%lf \n",dim,loopCount,tolerance);
     printf("time spent:%lf seconds \n",time_spent);  
    for(i=0;i<dim;i++)
      {
       if(i%10==0)
      {
       printf("\n");
       }
       printf("[%d]:%lf ",i,x[i]);
      }
      hipFree(cudaX); 
      hipFree(cudaM);
      
     // printf("The modified string: %s\n", a);
     return 0;
}

double r2()
{
    return (double)rand() / (double)RAND_MAX ;
}







