#include <stdio.h>
#include <hip/hip_runtime.h>

double r2();
__global__ void JORkernel(double *cudaM,int dim)
{
        //int idx = blockIdx.x * blockDim.x + threadIdx.x;        // Finds the thread_id
        int blocksize=16;
        int gridsize=gridDim.x; //this will always be 1
       // int gridsize2=gridDim.x;
       // printf("gridsize: %d %d\n", gridsize,gridsize2);
       
        int bx=blockIdx.x;
        int by=blockIdx.y;
        int tx=threadIdx.x;
        int ty=threadIdx.y;
        
        int blocktotal=blocksize*blocksize;
        int temp1=by*gridsize+bx;
        int temp2=ty*blocksize+tx;
       // printf("bx: %d \n",bx);
       // printf("by: %d \n",by);
       // printf("tx: %d \n",tx);
       // printf("ty: %d \n",ty);



       int ind=temp1*blocktotal+temp2;
       if(ind<dim)
     { 
        printf("%d \n",ind);
     }
}

int main()
{
        int i;
        int j;
        int dim=1000;
        double *matrix;
        matrix=(double*) malloc(dim*dim*sizeof(double));

        //the diagonal dominated matrix will be automatically be not singular!!
      for(i=0;i<dim;i++)
      {
        double rowSum=0.0;
        for(j=0;j<dim;j++)
        {
            
            matrix[i*dim+j]=r2();
            rowSum=matrix[i*dim+j]+rowSum;
        }
        matrix[i*dim+i]=rowSum;
      }
  //print the matrix out to check
     // for(i=0;i<dim;i++)
     // {
       // for(j=0;j<dim;j++)
      //  {
        //  printf("%lf ",matrix[i*dim+j]);
       // }
       // printf("\n");
     // }
      double *cudaM; //prepare for cuda global memory
      int msize=dim*dim*sizeof(double);
      hipMalloc((void**)&cudaM,msize);
      hipMemcpy(cudaM,matrix,msize,hipMemcpyHostToDevice);

     // cudaMalloc( (void**)&ad, csize );
     // cudaMalloc( (void**)&bd, isize );
     // cudaMemcpy( ad, a, csize, cudaMemcpyHostToDevice );
     // cudaMemcpy( bd, b, isize, cudaMemcpyHostToDevice );
      int blocksize=16;
      int gridsize= dim/256+1; //make the gridsize to be an ro  like (1,2)
      dim3 Grid( 1, gridsize);                   // Number of threads per block
      dim3 Block( blocksize,blocksize);              // Number of thread blocks
      JORkernel<<<Grid, Block>>>(cudaM,dim);
      //cudaMemcpy( a, ad, csize, cudaMemcpyDeviceToHost );
      hipFree( cudaM );
      
     // printf("The modified string: %s\n", a);
      return 0;
}

double r2()
{
    return (double)rand() / (double)RAND_MAX ;
}






