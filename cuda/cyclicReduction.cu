#include "hip/hip_runtime.h"
    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 9
   

    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int step,int powerNumber,int totalNumber,float** A, float** B, float** C, float** D)
    {
      //maybe have some way to enhance this, since some block don't need to load C and D
      //511 is getting from pow(2,EXPO-1)-1 and can be changed later.
      /*__shared__ float A_Local[511];
      __shared__ float B_Local[511];
      __shared__ float C_Local[511];
      __shared__ float D_Local[511];*/
      extern __shared__ float wholeArray[]; //dynamically allocate shared memory
      float* A_Local=(float*)&wholeArray[511];
      float* B_Local=(float*)&wholeArray[1022];
      float* C_Local=(float*)&wholeArray[1533];
      float* D_Local=(float*)&wholeArray[2044];

      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;
      int BLOCKSIZE=16;
      

      int temp=ty*BLOCKSIZE+tx;
      //need to notice threadId in different block should be the same
     /* for(int i=0;i<511;i++)
      {*/
        if(by!=1) //A has to be loaded in these blocks
        {
        A_Local[temp]=A[step-1][temp];
        }
        if(by!=0)
        {
         C_Local[temp]=C[step-1][temp];
        }
        if(by==3)
        {
         D_Local[temp]=D[step-1][temp];
        }
        //B need to be loaded for all the block, no if should apply to that
        B_Local[temp]=B[step-1][temp];
        __syncthreads();

            if(by==0)
            {
            for(int i=0;i<10;i++)
            {
                printf("cuda A: %f in step :%d \n", A_Local[i],step);
        
            }
        }


       if(by==0)//means this is the first block ,As will be calculated here
       {
        //if for boundary check
        if(temp-powerNumber>0)
        {
        A[step][temp]=(-1)*A_Local[temp]/(B_Local[temp-powerNumber])*A_Local[temp-powerNumber];
        }
        else
        {
         A[step][temp]=0;
        }
       }

       if(by==2) //means this is the third block, Cs will be calculated here
       {
        if(temp+powerNumber<totalNumber)
        {
         C[step][temp]=(-1)*C_Local[temp]/B_Local[temp+powerNumber]*C_Local[temp+powerNumber];   
        }
        else
        {
         C[step][temp]=0;
        }
       }

       if(by==1) //means this is the second block, Bs will be calculated here
       {
        if(temp-powerNumber>0 && temp+powerNumber<totalNumber)
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=totalNumber)
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<totalNumber)
        {
        B[step][temp]=B_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else
        {
        B[step][temp]=B_Local[temp];
        }
       }

       if(by==3) //this is the fourth block, Ds will be calculated here
       { 
        if(temp-powerNumber>0 && temp+powerNumber<totalNumber)
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=totalNumber)
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<totalNumber)
        {
        D[step][temp]=D_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else
        {
        D[step][temp]=D_Local[temp];
        }   
       }
      //}
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1;
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);
        /*int **by_global, **bx_global;*/
        /*float* A;
        float* B;
        float* C;
        float* D;

        A=(float*)malloc(m*sizeof(float));
        B=(float*)malloc(m*sizeof(float));
        C=(float*)malloc(m*sizeof(float));
        D=(float*)malloc(m*sizeof(float));*/

        float **A; //need a two dimension array to store different versin of A, so A will be A[step][i]; step is how many step will be 9 here and i will be 512 here.
        float **B;
        float **C;
        float **D;
       //each version j loop through 1 to n-1 and also the initial value so we need to 
        //remember EXPO of them
        //we need to remember them in order to use them later in back substitution


        A=(float**)malloc(EXPO*sizeof(float*));
        B=(float**)malloc(EXPO*sizeof(float*));
        C=(float**)malloc(EXPO*sizeof(float*));
        D=(float**)malloc(EXPO*sizeof(float*));

        for(int i=0;i<EXPO;i++)
        {
            A[i]=(float*)malloc(m*sizeof(float));
        }
         for(int i=0;i<EXPO;i++)
        {
            B[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            C[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            D[i]=(float*)malloc(m*sizeof(float));
        }

       //initialize A,B,C,D
        A[0][0]=0;
        for(int i=1;i<m;i++)
        {
            A[0][i]=1-delta*delta*0.5*i;
            if(i<10)
            {
                printf("%f \n",A[0][i]);
            }
        }
        for(int i=0;i<m;i++)
        {
            B[0][i]=-2+delta*delta*1.0;
        }
        C[0][m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[0][i]=1+0.5*delta*delta*i;
        }
        D[0][0]=2*pow(delta,3)-(1-0.5*delta*delta);
        for(int i=1;i<m;i++)
        {
            D[0][i]=2*(i+1)*pow(delta,3);
        }
        clock_t begin,end;
        begin=clock();
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        dim3 dimGrid(4,1); //so we have 4 blocks each block will in charge a,b,c,d respectly.
        dim3 dimBlock(16,16);

        //m is the size
        float ** AT,**BT,**CT,**DT;
        int size=m*sizeof(float*);

        hipMalloc((void**)&AT,size);
        hipMalloc((void**)&BT,size);
        hipMalloc((void**)&CT,size);
        hipMalloc((void**)&DT,size);

        hipMemcpy(AT,A,size,hipMemcpyHostToDevice);
        hipMemcpy(BT,B,size,hipMemcpyHostToDevice);
        hipMemcpy(CT,C,size,hipMemcpyHostToDevice);
        hipMemcpy(DT,D,size,hipMemcpyHostToDevice);

        printf("this is to test EXPO should see 9 here: %d \n",EXPO);

        for(int j=1;j<EXPO;j++)
        {
            //for each j do the work sequentially, inside this loop do work parallel.
          int powerNumber=pow(2,j-1);
          int totalNumber=m+1;
          //pass i the dynamically allocated shared memory among block.
           CalculatePArrayKernel<<<dimGrid,dimBlock,2044*sizeof(float)>>>(j,powerNumber,totalNumber,AT,BT,CT,DT);
           hipDeviceSynchronize();
           printf("called from host %d \n",j);
        }
        //copy data back to device
        hipMemcpy(A,AT,size,hipMemcpyDeviceToHost);
        hipMemcpy(B,BT,size,hipMemcpyDeviceToHost);
        hipMemcpy(C,CT,size,hipMemcpyDeviceToHost);
        hipMemcpy(D,DT,size,hipMemcpyDeviceToHost);
    
        double time_spent;


        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
        printf("time spend for 524 n points is :%f seconds \n",time_spent);

        for(int k=0;k<100;k++)
        {
         printf("A new 1: %f \n",A[1][k]);
          printf("A new 8: %f \n",A[8][k]);
        }
        
        hipFree(AT);
        hipFree(BT);
        hipFree(CT);
        hipFree(DT);
      //release memory
        for(int i=0;i<9;i++)
        {
            free(A[i]);
        }
        free(A);

        for(int i=0;i<9;i++)
        {
            free(B[i]);
        }
        free(B);

        for(int i=0;i<9;i++)
        {
            free(C[i]);
        }
        free(C);

        for(int i=0;i<9;i++)
        {
            free(D[i]);
        }
        free(D);

        return 0;
    }