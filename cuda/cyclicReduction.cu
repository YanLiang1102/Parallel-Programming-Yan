    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 3
   
    //the right way to add in cuda driver if you have an gpu
    //http://askubuntu.com/questions/451221/ubuntu-14-04-install-nvidia-driver


    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int totalStep,size_t pitch,float* A, float* B, float* C, float* D)
    {
      
      __shared__ float A_Local[7];
      __shared__ float B_Local[7];
      __shared__ float C_Local[7];
      __shared__ float D_Local[7];

      for(int step=1;step<totalStep;step++)
      {
      //this is a good graph to show how does cuda grid index working
      //http://stackoverflow.com/questions/26913683/different-way-to-index-threads-in-cuda-c
      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;
      //int BLOCKSIZE=16;
      int BLOCKSIZE=3;

    /*   if(tx==0&&ty==0)
        {  printf("step: %d has been called from : \%d! \n",step,by);
            
        }*/
      
      int totalNumber=(int) pow(2.0,totalStep*1.0);
      int columnCount=totalNumber-1;
      int powerNumber=(int) pow(2.0,step-1.0);
      int stopLoading=(int) (pow(2.0,totalStep*1.0)-pow(2.0,(step-1)*1.0));
      //according to the formula the stopLoading will stop load at 2^n-2^step, that is how we get this.
      

      int temp=ty*BLOCKSIZE+tx;
      int expoStep=(int)pow(2.0,(step-1)*1.0);

    if((temp<=stopLoading)&&(temp%(expoStep))==0)
    {
        if(by!=1) //A has to be loaded in these blocks
        {
        A_Local[temp]=A[(step-1)*columnCount+temp];
        }
        if(by!=0)
        {
         C_Local[temp]=C[(step-1)*columnCount+temp];
        }
        if(by==3)
        {
         D_Local[temp]=D[(step-1)*columnCount+temp];
        }
        //B need to be loaded for all the block, no if should apply to that
        B_Local[temp]=B[(step-1)*columnCount+temp];
        __syncthreads();

        //test A_Local
        if(by==2&&tx==0&&ty==0&&step==2)
        {
            printf("I should run only once for A_Local %d! \n",columnCount);
            for(int i=0;i<=stopLoading;i++)
            {
                printf("A local %f \n",A_Local[i]);
            }

        }

       if(by==0)//means this is the first block ,As will be calculated here
       {

       if(temp-powerNumber>0)
        {
        A[step*columnCount+temp]=(-1)*A_Local[temp]/(B_Local[temp-powerNumber])*A_Local[temp-powerNumber];
        }
        else
        {
         A[step*columnCount+temp]=0;
        }
        
       }

       if(by==2) //means this is the third block, Cs will be calculated here
       {
        if(temp+powerNumber<totalNumber)
        {
         C[step*columnCount+temp]=(-1)*C_Local[temp]/B_Local[temp+powerNumber]*C_Local[temp+powerNumber];   
        }
        else
        {
         C[step*columnCount+temp]=0;
        }
       }

       if(by==1) //means this is the second block, Bs will be calculated here
       {
        if(temp-powerNumber>0 && temp+powerNumber<totalNumber)
        {
        B[step*columnCount+temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=totalNumber)
        {
        B[step*columnCount+temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<totalNumber)
        {
        B[step*columnCount+temp]=B_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else
        {
        B[step*columnCount+temp]=B_Local[temp];
        }
       }

       if(by==3) //this is the fourth block, Ds will be calculated here
       { 
        if(temp-powerNumber>0 && temp+powerNumber<totalNumber)
        {
        D[step*columnCount+temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=totalNumber)
        {
        D[step*columnCount+temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<totalNumber)
        {
        D[step*columnCount+temp]=D_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else
        {
        D[step*columnCount+temp]=D_Local[temp];
        }   
       }
   }
       __syncthreads();
     }
      //}
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1;
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);

        float *A;
        float *B;
        float *C;
        float *D;

        int chunkSize=EXPO*m*sizeof(float);
        A=(float*)malloc(chunkSize);
        B=(float*)malloc(chunkSize);
        C=(float*)malloc(chunkSize);
        D=(float*)malloc(chunkSize);

        A[0]=0;
        //int vectorLength=EXPO*m;
        for(int i=1;i<=m;i++)
        {
            A[i]=1-delta*delta*0.5*i;
              if(i<=7)
            {
                printf("%f \n",A[i]);
            }
        }

        for(int i=0;i<m;i++)
        {
            B[i]=-2+delta*delta*1.0;
        }

        C[m-1]=0;
        for(int i=0;i<m;i++)
        {
            C[i]=1+0.5*delta*delta*i;
        }

        D[0]=0;
        for(int i=1;i<m;i++)
        {
            D[i]=2*(i+1)*pow(delta,3);
        }
        clock_t begin,end;
        begin=clock();
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        dim3 dimGrid(1,4); //so we have 4 blocks each block will in charge a,b,c,d respectly.
        dim3 dimBlock(3,3);

        //http://stackoverflow.com/questions/5029920/how-to-use-2d-arrays-in-cuda
        //according to the above post, the following is the correct way to allocate 2D array on cuda devixe

        float *deviceA, *deviceB, *deviceC, *deviceD;
        size_t pitch;
        hipMallocPitch((void**)&deviceA,&pitch,m*sizeof(float),EXPO);
        hipMallocPitch((void**)&deviceB,&pitch,m*sizeof(float),EXPO);
        hipMallocPitch((void**)&deviceC,&pitch,m*sizeof(float),EXPO);
        hipMallocPitch((void**)&deviceD,&pitch,m*sizeof(float),EXPO);


        int size=EXPO*m*sizeof(float);
        hipMemcpy(deviceA,A,size,hipMemcpyHostToDevice);
        hipMemcpy(deviceB,B,size,hipMemcpyHostToDevice);
        hipMemcpy(deviceC,C,size,hipMemcpyHostToDevice);
        hipMemcpy(deviceD,D,size,hipMemcpyHostToDevice);
        //deviceA, deviceB, deviceC, deviceD is designed to be the global memory of cuda.
        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(EXPO,pitch,deviceA,deviceB,deviceC,deviceD);

        hipMemcpy(A,deviceA,size,hipMemcpyDeviceToHost);
        hipMemcpy(B,deviceB,size,hipMemcpyDeviceToHost);
        hipMemcpy(C,deviceC,size,hipMemcpyDeviceToHost);
        hipMemcpy(D,deviceD,size,hipMemcpyDeviceToHost);
    
        double time_spent;


        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
        printf("time spend for 524 n points is :%f seconds \n",time_spent);
/*
        printf("hey here is the result matrix: \n");
        for(int k=0;k<EXPO*m;k++)
        {
            if(k%7==0)
         {
            printf("\n");
         }
         printf("%f ",B[k]);
         
        }*/
        
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        hipFree(deviceD);

        free(A);
        free(B);
        free(C);
        free(D);

        return 0;
    }