#include "hip/hip_runtime.h"
    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 9
    #define BLOCKSIZE 16

    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int step, float** A, float** B, float** C, float** D)
    {
      //maybe have some way to enhance this, since some block don't need to load C and D
      int local_dimension=pow(2,EXPO-1)-1
      __shared__ float A_Local[local_dimension];
      __shared__ float B_Local[local_dimension];
      __shared__ float C_Local[local_dimension];
      __shared__ float D_Local[local_dimension];

      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
          int ty=threadIdx.y;

      int temp=ty*BLOCKSIZE+tx;
      //need to notice threadId in different block should be the same
      for(int i=0;i<local_dimension;i++)
      {
        if(by!=1) //A has to be loaded in these blocks
        {
        A_Local[temp]=A[step-1][temp];
        }
        if(by!=0)
        {
         C_Local[temp]=C[step-1][temp];
        }
        if(by==3)
        {
         D_Local[temp]=D[step-1][temp];
        }
        //B need to be loaded for all the block, no if should apply to that
         B_Local[temp]=B[step-1][temp];
        __syncthreads();

       if(by==0)//means this is the first block ,As will be calculated here
       {
        //if for boundary check
        if(temp-pow(2,step-1)>0)
        {
        A[step][temp]=(-1)*A_Local[temp]/(B_Local[temp-pow(2,step-1)])*A_Local[temp-pow(2,step-1)];
        }
        else
        {
         A[step][temp]=0;
        }
       }

       if(by==2) //means this is the third block, Cs will be calculated here
       {
        if(temp+pow(2,step-1)<pow(2,EXPO))
        {
         C[step][temp]=(-1)*C_Local[temp]/B_Local[temp+pow(2,step-1)]*C_Local[temp+pow(2,step-1)];   
        }
        else
        {
         C[step][temp]=0;
        }
        }

       if(by==1) //means this is the second block, Bs will be calculated here
       {
        if(temp-pow(2,step-1)>0 && temp+pow(2,step-1)<pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-pow(2,step-1)]*C_Local[temp-pow(2,step-1)]-C_Local[temp]/B_Local[temp+pow(2,step-1)]*A_Local[temp+pow(2,step-1)];
        }
        else if(temp-pow(2,step-1)>0 && temp+pow(2,step-1)>=pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-pow(2,step-1)]*C_Local[temp-pow(2,step-1)];
        }
        else if(temp-pow(2,step-1)<=0 && temp+pow(2,step-1)<pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-C_Local[temp]/B_Local[temp+pow(2,step-1)]*A_Local[temp+pow(2,step-1)];
        }
        else
        {
        B[step][temp]=B_Local[temp];
        }
       }

       if(by==3) //this is the fourth block, Ds will be calculated here
       { 
        if(temp-pow(2,step-1)>0 && temp+pow(2,step-1)<pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-pow(2,step-1)]*D_Local[temp-pow(2,step-1)]-C_Local[temp]/B_Local[temp+pow(2,step-1)]*D_Local[temp+pow(2,step-1)]; 
        }
        else if(temp-pow(2,step-1)>0 && temp+pow(2,step-1)>=pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-pow(2,step-1)]*D_Local[temp-pow(2,step-1)];
        }
        else if(temp-pow(2,step-1)<=0 && temp+pow(2,step-1)<pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-C_Local[temp]/B_Local[temp+pow(2,step-1)]*D_Local[temp+pow(2,step-1)]; 
        }
        else
        {
        D[step][temp]=D_Local[temp];
        }   
       }
      }
    }

    int main()
    {
    	
        int m=pow(2,EXPO)-1;
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);
        /*float* A;
        float* B;
        float* C;
        float* D;

        A=(float*)malloc(m*sizeof(float));
        B=(float*)malloc(m*sizeof(float));
        C=(float*)malloc(m*sizeof(float));
        D=(float*)malloc(m*sizeof(float));*/

        float **A; //need a two dimension array to store different versin of A
        float **B;
        float **C;
        float **D;
       //each version j loop through 1 to n-1 and also the initial value so we need to 
        //remember EXPO of them
        //we need to remember them in order to use them later in back substitution


        A=(float**)malloc(EXPO*sizeof(float*));
        B=(float**)malloc(EXPO*sizeof(float*));
        C=(float**)malloc(EXPO*sizeof(float*));
        D=(float**)malloc(EXPO*sizeof(float*));

        for(int i=0;i<EXPO;i++)
        {
            A[i]=(float*)malloc(m*sizeof(float));
        }
         for(int i=0;i<EXPO;i++)
        {
            B[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            C[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            D[i]=(float*)malloc(m*sizeof(float));
        }

       //initialize A,B,C,D
        A[0][0]=0;
        for(int i=1;i<m;i++)
        {
            A[0][i]=1-delta*delta*0.5*i;
            if(i<10)
            {
                printf("%f \n",A[0][i]);
            }
        }
        for(int i=0;i<m;i++)
        {
            B[0][i]=-2+delta*delta*1.0;
        }
        C[0][m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[0][i]=1+0.5*delta*delta*i;
        }
        D[0][0]=2*pow(delta,3)-(1-0.5*delta*delta);
        for(int i=1;i<m;i++)
        {
            D[0][i]=2*(i+1)*pow(delta,3);
        }
        return 0;
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        dim3 dimGrid(4,1); //so we have 4 blocks each block will in charge a,b,c,d respectly.
        dim3 dimBlock(16,16);

        for(int j=1;j<EXPO;j++)
        {
        	//for each j do the work sequentially, inside this loop do work parallel.
          
           CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j);
        }


    }