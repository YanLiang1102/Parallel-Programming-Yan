    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 3
   
    //the right way to add in cuda driver if you have an gpu
    //http://askubuntu.com/questions/451221/ubuntu-14-04-install-nvidia-driver


    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int totalStep,int step,int blockRow, int blockColumn,float* deviceA, float* deviceB, float* deviceC, float* deviceD)
    {
      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;

     //thread (tx,ty) should work on item helper3 in A,B,C,D

    //for (i=1;i<=2^(EXPO-step)-1;i++)
     //the last processor will not be used so check
      int helper11=pow(2.0,(EXPO+1)*1.0);
    
 
      int helper22=pow(2.0,(EXPO-step+1)*1.0);
        int helper33=pow(2.0,EXPO*1.0)-1;
        printf("step is running: %d \n",step);

     // if(helper3<pow(2.0,(EXPO-step)*1.0)-1)
        //step 1 is special case.
       if(ty!=blockColumn-1&&tx!=blockRow-1)
      {
        // in the formula i will be i=helper3
        /*****calcualte A******************/
        int helper1=helper11;
        int helper2=helper22;
        int flag=0;//special for step1.
        if(step==1)
        {
            helper1=0;
            helper2=0;
            flag=1;
        }

        int helper3=ty*blockColumn+tx+1;
        float ahelperfora1=deviceA[-step+helper1-helper2+2*(helper3)];
        float ahelperfora2=deviceA[-step+helper1-helper2+2*(helper3)-1];
        float bhelperfora1=deviceB[-step+helper1-helper2+2*(helper3)-1];
        deviceA[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*(ahelperfora1)*ahelperfora2/bhelperfora1;

         if(step==1&&tx==1&&ty==0)
         {
            printf("let me see flag: %d \n", flag);
            printf("helper1:%d \n",helper1);
            printf("helper2: %d \n",helper2);
             printf("helper3: %d \n",helper3);
 /*           printf("deviceA[0]:%f \n",deviceA[0]);
              printf("deviceA[1]:%f \n",deviceA[1]);
                printf("deviceB[0]:%f \n",deviceB[0]);*/
            printf("index is :%d \n",-step+helper1-helper2+2*(helper3));
            printf("ahelperfora1 is %f \n",ahelperfora1);
              printf("ahelperfora2 is %f \n",ahelperfora2);
              printf("bhelperfora1 is %f \n",bhelperfora1);
              printf("give me the result: %f",-1*(ahelperfora1)*ahelperfora2/bhelperfora1);
              printf("which one you are calculate tehre? %d \n",-1-step+helper1-helper2+helper3+flag*(1+helper33));


         }

        //*****calculate C******************/
        float chelperforc1=deviceC[-step+helper1-helper2+2*(helper3)];
        float chelperforc2=deviceC[-step+helper1-helper2+2*(helper3)+1];
        float bhelperforc2=deviceB[-step+helper1-helper2+2*(helper3)+1];
        deviceC[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*chelperforc1*chelperforc2/bhelperforc2;

        //calculate B***********************************************//
        float bhelperforb1=deviceB[-step+helper1-helper2+2*(helper3)];
        float bhelperforb2=deviceB[-step+helper1-helper2+2*(helper3)-1];
        float bhelperforb3=deviceB[-step+helper1-helper2+2*(helper3)+1];
        float ahelperforb1=deviceA[-step+helper1-helper2+2*(helper3)];
        float ahelperforb2=deviceA[-step+helper1-helper2+2*(helper3)+1];
        float chelperforb1=deviceC[-step+helper1-helper2+2*(helper3)-1];
        float chelperforb2=deviceC[-step+helper1-helper2+2*(helper3)];
        deviceB[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=bhelperforb1-ahelperforb1/bhelperforb2*chelperforb1-chelperforb2/bhelperforb3*ahelperforb2;

        //calculate D***************************************************//
        float dhelperford1=deviceD[-step+helper1-helper2+2*(helper3)];
        float dhelperford2=deviceD[-step+helper1-helper2+2*(helper3)-1];
        float dhelperford3=deviceD[-step+helper1-helper2+2*(helper3)+1];
        float ahelperford1=deviceA[-step+helper1-helper2+2*(helper3)];
        float bhelperford1=deviceB[-step+helper1-helper2+2*(helper3)-1];
        float bhelperford2=deviceB[-step+helper1-helper2+2*(helper3)+1];
        float chelperford1=deviceC[-step+helper1-helper2+2*(helper3)];
        deviceD[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=dhelperford1-ahelperford1/bhelperford1*dhelperford2-chelperford1/bhelperford2*dhelperford3;
          if(step==1&&tx==0&&ty==0)
      {
        for (int i=0;i<7;i++)
        {
            printf("deviceA in step1: %f \n",deviceA[i]);
         
        }
        for (int i=0;i<7;i++)
        {
            printf("deviceB in step 1: %f \n",deviceB[i]);
         
        }
        for (int i=0;i<7;i++)
        {
            printf("deviceC in step 1: %f \n",deviceC[i]);
         
        }
        for (int i=0;i<7;i++)
        {
            printf("deviceD in step 1: %f \n",deviceD[i]);
         
        }
      }
       if(step==2&&tx==0&&ty==0)
      {
        for (int i=0;i<10;i++)
        {
            printf("deviceA in step2: %f \n",deviceA[i]);
         
        }
        for (int i=0;i<10;i++)
        {
            printf("deviceB in step 2: %f \n",deviceB[i]);
         
        }
        for (int i=0;i<10;i++)
        {
            printf("deviceC in step 2: %f \n",deviceC[i]);
         
        }
        for (int i=0;i<10;i++)
        {
            printf("deviceD in step 2: %f \n",deviceD[i]);
         
        }
      }
       
      
    }
    __syncthreads();
      
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1; //think of our example as n=3 then m will be 7 here
        printf("m value is %d",m);
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);  //this is correct , think of m as the number of inner 

        float *A;
        float *B;
        float *C;
        float *D;

        //by careful calculation, we figure out we need (2^n-1)*2
        //so the orinal step need to store 2^n-1 value, then step 1 needs 2^(n-1)-1 value and the last one will be 2^1-1 value.
        //so chuck size will be 2^n-1+2^(n-1)-1+....+2-1
        int chunkLength=(pow(2,EXPO)-1)*2;
        int chunkSize=chunkLength*sizeof(float);
        A=(float*)malloc(chunkSize);
        B=(float*)malloc(chunkSize);
        C=(float*)malloc(chunkSize);
        D=(float*)malloc(chunkSize);

        A[0]=0;
        //int vectorLength=EXPO*m;
        printf("m value is %d",m);
        for(int i=1;i<m;i++)
        {
            A[i]=1-delta*delta*0.5*(i+1);
              if(i<7)
            {
                printf("%f \n",A[i]);
            }
                //printf("m value: %d",m);
        }
        //else will be 0
        for(int i=m;i<chunkLength;i++)
        {
          A[i]=0;
        }
        printf("fail here?");
  
        printf("maybe index out of range?");
       

        for(int i=0;i<m;i++)
        {
            B[i]=-2+delta*delta*1.0;
        }
          for(int i=m;i<chunkLength;i++)
        {
          B[i]=0;
        }

        C[m-1]=0;
        for(int i=0;i<m;i++)
        {
            C[i]=1+0.5*delta*delta*(i+1);
        }
        for(int i=m;i<chunkLength;i++)
        {
          C[i]=0;
        }


        D[0]=2*pow(delta,3)+0.5*delta*delta-1;
        for(int i=1;i<m;i++)
        {
            D[i]=2*(i+1)*pow(delta,3);
        }
        for(int i=m;i<chunkLength;i++)
        {
          D[i]=0;
        }


        clock_t begin,end;
        begin=clock();
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        //dim3 dimGrid(1,4); //so we have 4 blocks each block will in charge a,b,c,d respectly.

   

        //http://stackoverflow.com/questions/5029920/how-to-use-2d-arrays-in-cuda
        //according to the above post, the following is the correct way to allocate 2D array on cuda devixe

    /*    float *deviceA, *deviceB, *deviceC, *deviceD;
        size_t pitch;
        cudaMallocPitch((void**)&deviceA,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceB,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceC,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceD,&pitch,m*sizeof(float),EXPO);*/
        float *deviceA, *deviceB, *deviceC, *deviceD;
        hipMalloc((void**)&deviceA,chunkSize);
        hipMalloc((void**)&deviceB,chunkSize);
        hipMalloc((void**)&deviceC,chunkSize);
        hipMalloc((void**)&deviceD,chunkSize);
        printf("is it still working after cuda Malloc??\n");
        //copy the host vector to device.
        hipMemcpy(deviceA,A,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceD,D,chunkSize,hipMemcpyHostToDevice);
        //deviceA, deviceB, deviceC, deviceD is designed to be the global memory of cuda.
        for(int j=1;j<EXPO;j++)
        {
        //the lock size should change, the first step it will need 2^(n-j)-1, so first step will be 3 if n=3
        dim3 dimGrid(1,1);
        int blockRow=pow(2,(EXPO-j)/2);
        printf("blockrow is :%d \n",blockRow);
        int blockColumn=pow(2,EXPO-j-(EXPO-j)/2);
        printf("blockColumn is :%d \n",blockColumn);
        dim3 dimBlock(blockColumn,blockRow);
          //in each step the processor being used should decrease should be 2^(n-j)-1 in jth step
        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(EXPO,j,blockRow,blockColumn,deviceA,deviceB,deviceC,deviceD);


        }
       /* dim3 dimGrid(1,1);
        //int blockRow=pow(2,(EXPO-1)/2);
        //int blockColumn=pow(2,EXPO-1-(EXPO-1)/2);
        dim3 dimBlock(2,2);
        printf("did you run here at least?");
        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(EXPO,2,2,deviceA,deviceB,deviceC,deviceD);*/

        
        //copy the device vector to host
        hipMemcpy(A,deviceA,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(B,deviceB,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(C,deviceC,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(D,deviceD,chunkSize,hipMemcpyDeviceToHost);
        
    
        double time_spent;


        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
        printf("time is :%f seconds \n",time_spent);

        
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        hipFree(deviceD);

        free(A);
        free(B);
        free(C);
        free(D);

        return 0;
    }