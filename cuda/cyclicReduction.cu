    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 9
   
    //the right way to add in cuda driver if you have an gpu
    //http://askubuntu.com/questions/451221/ubuntu-14-04-install-nvidia-driver




     __global__ void BackwardKernel(int k,int blockRow, int blockColumn,float* deviceA, float* deviceB, float* deviceC, float* deviceD,float* deviceFinalX,float initialValue)
     {
      int bx1=blockIdx.x;
      int by1=blockIdx.y;
      int tx1=threadIdx.x;
      int ty1=threadIdx.y;
      //printf("inside of kernle %f \n",deviceFinalX[4]);

      int backhelper1=ty1*blockColumn+tx1+1;
      int backhelper2=2*backhelper1-1;//(int((2*backhelper1-1)*pow(2.0,1.0*(k-1))))/(int)(pow(2.0,(k-1)*1.0));
      int backhelper3=(int)pow(2.0,(EXPO+1)*1.0);
      int backhelper4=(int)pow(2.0,(EXPO-k+2)*1.0);


      int h=(int)(pow(2.0,1.0*(k-1)));

      float backhelperd=deviceD[-k+backhelper3-backhelper4+backhelper2];
      float backhelpera=deviceA[-k+backhelper3-backhelper4+backhelper2];
      float backhelperb=deviceB[-k+backhelper3-backhelper4+backhelper2];
      float backhelperc=deviceC[-k+backhelper3-backhelper4+backhelper2];

      int xindex1=backhelper2*pow(2.0,1.0*(k-1))-h;
      int xindex2=backhelper2*pow(2.0,1.0*(k-1))+h;

      //so thread i will be in charge of (2i-1)*2^(k-1) calculation
      //printf("%d ",int((2*backhelper1-1)*pow(2.0,1.0*(k-1))));
      deviceFinalX[(int)(backhelper2*pow(2.0,1.0*(k-1)))]=(backhelperd-backhelpera*deviceFinalX[xindex1]-backhelperc*deviceFinalX[xindex2])*1.0/backhelperb;

      __syncthreads();
     }

    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int step,int blockRow, int blockColumn,float* deviceA, float* deviceB, float* deviceC, float* deviceD)
    {
      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;

      int helper11=pow(2.0,(EXPO+1)*1.0);
      int helper22=pow(2.0,(EXPO-step+1)*1.0);
      int helper44=pow(2.0,(EXPO-step+2)*1.0);
      int helper33=pow(2.0,EXPO*1.0)-1;
        //printf("step is running: %d \n",step);

     // if(helper3<pow(2.0,(EXPO-step)*1.0)-1)
        //--step 1 is special case.
     /*  if((tx!=(blockColumn-1))&&(ty!=(blockRow-1)))-----this is very important branch divergence happen here, need
     //to figure out how exactly cuda works!!
        /*****calcualte A******************/
        int helper1=helper11;
        int helper2=helper22;
        int helper4=helper44;
        int flag=0;//special for step1.
        if(step==1)
        {
            helper1=0;
            helper2=0;
            helper4=0;
            flag=1;
        }

        int helper3=ty*blockColumn+tx+1;
        if(helper3<=(pow(2.0,1.0*(EXPO-step))-1.0))
        {
        float ahelperfora1=deviceA[-step+helper1-helper4+2*(helper3)];
        float ahelperfora2=deviceA[-step+helper1-helper4+2*(helper3)-1];
        float bhelperfora1=deviceB[-step+helper1-helper4+2*(helper3)-1];
        deviceA[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*(ahelperfora1)*ahelperfora2/bhelperfora1;

        //*****calculate C******************/
        float chelperforc1=deviceC[-step+helper1-helper4+2*(helper3)];
        float chelperforc2=deviceC[-step+helper1-helper4+2*(helper3)+1];
        float bhelperforc1=deviceB[-step+helper1-helper4+2*(helper3)+1];
        deviceC[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*chelperforc1*chelperforc2/bhelperforc1;

        //calculate B***********************************************//
        float bhelperforb1=deviceB[-step+helper1-helper4+2*(helper3)];
        float bhelperforb2=deviceB[-step+helper1-helper4+2*(helper3)-1];
        float bhelperforb3=deviceB[-step+helper1-helper4+2*(helper3)+1];
        float ahelperforb1=deviceA[-step+helper1-helper4+2*(helper3)];
        float ahelperforb2=deviceA[-step+helper1-helper4+2*(helper3)+1];
        float chelperforb1=deviceC[-step+helper1-helper4+2*(helper3)-1];
        float chelperforb2=deviceC[-step+helper1-helper4+2*(helper3)];
        deviceB[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=bhelperforb1-ahelperforb1/bhelperforb2*chelperforb1-chelperforb2/bhelperforb3*ahelperforb2;

        //calculate D***************************************************//
        float dhelperford1=deviceD[-step+helper1-helper4+2*(helper3)];
        float dhelperford2=deviceD[-step+helper1-helper4+2*(helper3)-1];
        float dhelperford3=deviceD[-step+helper1-helper4+2*(helper3)+1];
        float ahelperford1=deviceA[-step+helper1-helper4+2*(helper3)];
        float bhelperford1=deviceB[-step+helper1-helper4+2*(helper3)-1];
        float bhelperford2=deviceB[-step+helper1-helper4+2*(helper3)+1];
        float chelperford1=deviceC[-step+helper1-helper4+2*(helper3)];
        deviceD[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=dhelperford1-ahelperford1/bhelperford1*dhelperford2-chelperford1/bhelperford2*dhelperford3;
    }

        __syncthreads();
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1; //think of our example as n=3 then m will be 7 here
        /*printf("m value is %d",m);*/
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);  //this is correct , think of m as the number of inner 

        float *A;
        float *B;
        float *C;
        float *D;
        float *FinalX;

        //by careful calculation, we figure out we need (2^n-1)*2
        //so the orinal step need to store 2^n-1 value, then step 1 needs 2^(n-1)-1 value and the last one will be 2^1-1 value.
        //so chuck size will be 2^n-1+2^(n-1)-1+....+2-1
        //int chunkLength=(pow(2,EXPO)-1)*2;
        //ad one for the extra thread that never going to use, so in this way it will not be out of index
        int finalLengthX=(int)pow(2,EXPO)+1;
        int chunkLength=(pow(2,EXPO)-1)*2+1;
        int chunkSize=chunkLength*sizeof(float);
        A=(float*)malloc(chunkSize);
        B=(float*)malloc(chunkSize);
        C=(float*)malloc(chunkSize);
        D=(float*)malloc(chunkSize);
        FinalX=(float*)malloc(finalLengthX*sizeof(float));

        A[0]=0;
        //int vectorLength=EXPO*m;
     
        for(int i=1;i<m;i++)
        {
            A[i]=1-delta*delta*0.5*(i+1);
        }
        //else will be 0
        for(int i=m;i<chunkLength;i++)
        {
          A[i]=0;
        }

        for(int i=0;i<m;i++)
        {
            B[i]=-2+delta*delta*1.0;
        }
          for(int i=m;i<chunkLength;i++)
        {
          B[i]=0;
        }

        C[m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[i]=1+0.5*delta*delta*(i+1);
        }
        for(int i=m;i<chunkLength;i++)
        {
          C[i]=0;
        }


       /* D[0]=2*delta*delta*delta+0.5*delta*delta-1;*/
        for(int i=0;i<m-1;i++)
        {
            D[i]=2*(i+1)*pow(delta,3);
        }
        D[m-1]=2*m*delta*delta*delta-1+3.5*delta*delta;
        for(int i=m;i<chunkLength;i++)
        {
          D[i]=0;
        }


        clock_t begin,end;
        begin=clock();
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        //dim3 dimGrid(1,4); //so we have 4 blocks each block will in charge a,b,c,d respectly.

   

        //http://stackoverflow.com/questions/5029920/how-to-use-2d-arrays-in-cuda
        //according to the above post, the following is the correct way to allocate 2D array on cuda devixe

    /*    float *deviceA, *deviceB, *deviceC, *deviceD;
        size_t pitch;
        cudaMallocPitch((void**)&deviceA,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceB,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceC,&pitch,m*sizeof(float),EXPO);
        cudaMallocPitch((void**)&deviceD,&pitch,m*sizeof(float),EXPO);*/

        float *deviceA, *deviceB, *deviceC, *deviceD,*deviceFinalX;
        hipMalloc((void**)&deviceA,chunkSize);
        hipMalloc((void**)&deviceB,chunkSize);
        hipMalloc((void**)&deviceC,chunkSize);
        hipMalloc((void**)&deviceD,chunkSize);
        hipMalloc((void**)&deviceFinalX,finalLengthX*sizeof(float));

       
        //copy the host vector to device.
        hipMemcpy(deviceA,A,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceD,D,chunkSize,hipMemcpyHostToDevice);
        //deviceA, deviceB, deviceC, deviceD is designed to be the global memory of cuda.
        //forward
        for(int j=1;j<EXPO;j++)
        {
        //the lock size should change, the first step it will need 2^(n-j)-1, so first step will be 3 if n=3
        dim3 dimGrid(1,1);
        int blockRow=pow(2,(EXPO-j)/2);
        //printf("blockrow is :%d \n",blockRow);
        int blockColumn=pow(2,EXPO-j-(EXPO-j)/2);
        //printf("blockColumn is :%d \n",blockColumn);
        dim3 dimBlock(blockColumn,blockRow);
          //in each step the processor being used should decrease should be 2^(n-j)-1 in jth step
        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j,blockRow,blockColumn,deviceA,deviceB,deviceC,deviceD);

        }


        //backward
        //copy the device vector to host
        hipMemcpy(A,deviceA,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(B,deviceB,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(C,deviceC,chunkSize,hipMemcpyDeviceToHost);
        hipMemcpy(D,deviceD,chunkSize,hipMemcpyDeviceToHost);
        int lastIndex=(int)pow(2,EXPO+1)-EXPO-3;
        float initialValue=D[lastIndex]/B[lastIndex];
        FinalX[0]=0;
        FinalX[(int)pow(2,EXPO-1)]=initialValue;
        printf("the value in the middle is: %f and this suppose to close to 0.5 when n goes big! \n",FinalX[(int)pow(2,EXPO-1)]);

         hipMemcpy(deviceFinalX,FinalX,finalLengthX*sizeof(float),hipMemcpyHostToDevice);
        for(int k=EXPO-1;k>=1;k--)
        {
          //so the most one will use 2^(n-k) variable will be covered!
        dim3 dimGrid(1,1);
        int blockRow=pow(2,(EXPO-k)/2);
        int blockColumn=pow(2,EXPO-k-(EXPO-k)/2);
        dim3  dimBlock(blockColumn,blockRow);
        
        BackwardKernel<<<dimGrid,dimBlock>>>(k,blockRow,blockColumn,deviceA,deviceB,deviceC,deviceD,deviceFinalX,initialValue);


        }


        hipMemcpy(FinalX,deviceFinalX,finalLengthX*sizeof(float),hipMemcpyDeviceToHost);
          printf(" \n");
          printf(" A \n");
        for(int i=0;i<chunkLength;i++)
        {   
            if(i%8==0)
            {
                printf("\n");
            }
            printf("%f ",A[i]);
        }
             printf(" \n");
            printf(" B \n");
        for(int i=0;i<chunkLength;i++)
        {
              if(i%8==0)
            {
                printf("\n");
            }
            printf("%f ",B[i]);
        }
            printf(" \n");
            printf(" C \n");
        for(int i=0;i<chunkLength;i++)
        {
              if(i%8==0)
            {
                printf("\n");
            }
            printf("%f ",C[i]);
        }
            printf(" \n");
            printf(" D \n");
        for(int i=0;i<chunkLength;i++)
        {
              if(i%8==0)
            {
                printf("\n");
            }
            printf("%f ",D[i]);
        }

        
        double time_spent;


        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
   
          
          printf("\n the following are the solutions.");
          for(int i=0;i<finalLengthX;i++)
          {
            if(i%8==0)
            {
                printf("\n");
            }
            printf("%f ",FinalX[i]);
          }
        printf("\n time used to calculate this is :%f seconds \n",time_spent);
       
        
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        hipFree(deviceD);

        free(A);
        free(B);
        free(C);
        free(D);

        return 0;
    }
