    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 9
   

    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int step, float** A, float** B, float** C, float** D)
    {
      //maybe have some way to enhance this, since some block don't need to load C and D
      int local_dimension=pow(2,EXPO-1)-1;
      __shared__ float A_Local[511];
      __shared__ float B_Local[511];
      __shared__ float C_Local[511];
      __shared__ float D_Local[511];

      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;
      int BLOCKSIZE=16;
      int powerNumber=(int)(pow(2.0,(step-1)*1.0));

      int temp=ty*BLOCKSIZE+tx;
      //need to notice threadId in different block should be the same
      for(int i=0;i<local_dimension;i++)
      {
        if(by!=1) //A has to be loaded in these blocks
        {
        A_Local[temp]=A[step-1][temp];
        }
        if(by!=0)
        {
         C_Local[temp]=C[step-1][temp];
        }
        if(by==3)
        {
         D_Local[temp]=D[step-1][temp];
        }
        //B need to be loaded for all the block, no if should apply to that
         B_Local[temp]=B[step-1][temp];
        __syncthreads();

       if(by==0)//means this is the first block ,As will be calculated here
       {
        //if for boundary check
        if(temp-powerNumber>0)
        {
        A[step][temp]=(-1)*A_Local[temp]/(B_Local[temp-powerNumber])*A_Local[temp-powerNumber];
        }
        else
        {
         A[step][temp]=0;
        }
       }

       if(by==2) //means this is the third block, Cs will be calculated here
       {
        if(temp+powerNumber<pow(2,EXPO))
        {
         C[step][temp]=(-1)*C_Local[temp]/B_Local[temp+powerNumber]*C_Local[temp+powerNumber];   
        }
        else
        {
         C[step][temp]=0;
        }
        }

       if(by==1) //means this is the second block, Bs will be calculated here
       {
        if(temp-powerNumber>0 && temp+powerNumber<pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*C_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<pow(2,EXPO))
        {
        B[step][temp]=B_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*A_Local[temp+powerNumber];
        }
        else
        {
        B[step][temp]=B_Local[temp];
        }
       }

       if(by==3) //this is the fourth block, Ds will be calculated here
       { 
        if(temp-powerNumber>0 && temp+powerNumber<pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else if(temp-powerNumber>0 && temp+powerNumber>=pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-A_Local[temp]/B_Local[temp-powerNumber]*D_Local[temp-powerNumber];
        }
        else if(temp-powerNumber<=0 && temp+powerNumber<pow(2,EXPO))
        {
        D[step][temp]=D_Local[temp]-C_Local[temp]/B_Local[temp+powerNumber]*D_Local[temp+powerNumber]; 
        }
        else
        {
        D[step][temp]=D_Local[temp];
        }   
       }
      }
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1;
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);
        /*float* A;
        float* B;
        float* C;
        float* D;

        A=(float*)malloc(m*sizeof(float));
        B=(float*)malloc(m*sizeof(float));
        C=(float*)malloc(m*sizeof(float));
        D=(float*)malloc(m*sizeof(float));*/

        float **A; //need a two dimension array to store different versin of A
        float **B;
        float **C;
        float **D;
       //each version j loop through 1 to n-1 and also the initial value so we need to 
        //remember EXPO of them
        //we need to remember them in order to use them later in back substitution


        A=(float**)malloc(EXPO*sizeof(float*));
        B=(float**)malloc(EXPO*sizeof(float*));
        C=(float**)malloc(EXPO*sizeof(float*));
        D=(float**)malloc(EXPO*sizeof(float*));

        for(int i=0;i<EXPO;i++)
        {
            A[i]=(float*)malloc(m*sizeof(float));
        }
         for(int i=0;i<EXPO;i++)
        {
            B[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            C[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            D[i]=(float*)malloc(m*sizeof(float));
        }

       //initialize A,B,C,D
        A[0][0]=0;
        for(int i=1;i<m;i++)
        {
            A[0][i]=1-delta*delta*0.5*i;
            if(i<10)
            {
                printf("%f \n",A[0][i]);
            }
        }
        for(int i=0;i<m;i++)
        {
            B[0][i]=-2+delta*delta*1.0;
        }
        C[0][m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[0][i]=1+0.5*delta*delta*i;
        }
        D[0][0]=2*pow(delta,3)-(1-0.5*delta*delta);
        for(int i=1;i<m;i++)
        {
            D[0][i]=2*(i+1)*pow(delta,3);
        }
       clock_t begin,end;
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        dim3 dimGrid(4,1); //so we have 4 blocks each block will in charge a,b,c,d respectly.
        dim3 dimBlock(16,16);

        //m is the size
        float ** AT,**BT,**CT,**DT;
        hipMalloc((void***)&AT,m);
        hipMalloc((void***)&BT,m);
        hipMalloc((void***)&CT,m);
        hipMalloc((void***)&DT,m);

        hipMemcpy(AT,A,m,hipMemcpyHostToDevice);

        hipMemcpy(BT,B,m,hipMemcpyHostToDevice);
        hipMemcpy(CT,C,m,hipMemcpyHostToDevice);
        hipMemcpy(DT,D,m,hipMemcpyHostToDevice);

        for(int j=1;j<EXPO;j++)
        {
            //for each j do the work sequentially, inside this loop do work parallel.
          
           CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j,AT,BT,CT,DT);
        }
        //copy data back to device
        hipMemcpy(A,AT,m,hipMemcpyDeviceToHost);
        hipMemcpy(B,BT,m,hipMemcpyDeviceToHost);
        hipMemcpy(C,CT,m,hipMemcpyDeviceToHost);
        hipMemcpy(D,DT,m,hipMemcpyDeviceToHost);
    
        double time_spent;

        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
        printf("time spend for 524 n points is :%f",time_spent);

        for(int k=0;k<10;k++)
        {
         printf("A new: %f \n",A[1][k]);
        }
        
        hipFree(AT);
        hipFree(BT);
        hipFree(CT);
        hipFree(DT);

        return 0;
    }