#include "hip/hip_runtime.h"
    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 9
   

    //this is the kernel to calculate the P=(a,b,c,d)
    //need to pass in the step which is j, and then figure out which thread to work on
    //the calculation in (2^j,2*2^j,3*2^j....)
    __global__ void CalculatePArrayKernel(int step,int powerNumber,int totalNumber,float** A, float** B, float** C, float** D)
    {
      //maybe have some way to enhance this, since some block don't need to load C and D
      //511 is getting from pow(2,EXPO-1)-1 and can be changed later.
    

      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;
     
      

      int temp=ty*BLOCKSIZE+tx;
       
      printf("hello from step \n",step);
      //}
    }

    int main()
    {
        
        int m=pow(2,EXPO)-1;
        int b=1;
        int a=0;
        float delta=(b-a)*1.0/(m+1.0);
        /*int **by_global, **bx_global;*/
        /*float* A;
        float* B;
        float* C;
        float* D;

        A=(float*)malloc(m*sizeof(float));
        B=(float*)malloc(m*sizeof(float));
        C=(float*)malloc(m*sizeof(float));
        D=(float*)malloc(m*sizeof(float));*/

        float **A; //need a two dimension array to store different versin of A, so A will be A[step][i]; step is how many step will be 9 here and i will be 512 here.
        float **B;
        float **C;
        float **D;
       //each version j loop through 1 to n-1 and also the initial value so we need to 
        //remember EXPO of them
        //we need to remember them in order to use them later in back substitution


        A=(float**)malloc(EXPO*sizeof(float*));
        B=(float**)malloc(EXPO*sizeof(float*));
        C=(float**)malloc(EXPO*sizeof(float*));
        D=(float**)malloc(EXPO*sizeof(float*));

        for(int i=0;i<EXPO;i++)
        {
            A[i]=(float*)malloc(m*sizeof(float));
        }
         for(int i=0;i<EXPO;i++)
        {
            B[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            C[i]=(float*)malloc(m*sizeof(float));
        }
        for(int i=0;i<EXPO;i++)
        {
            D[i]=(float*)malloc(m*sizeof(float));
        }

       //initialize A,B,C,D
        A[0][0]=0;
        for(int i=1;i<m;i++)
        {
            A[0][i]=1-delta*delta*0.5*i;
            if(i<10)
            {
                printf("%f \n",A[0][i]);
            }
        }
        for(int i=0;i<m;i++)
        {
            B[0][i]=-2+delta*delta*1.0;
        }
        C[0][m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[0][i]=1+0.5*delta*delta*i;
        }
        D[0][0]=2*pow(delta,3)-(1-0.5*delta*delta);
        for(int i=1;i<m;i++)
        {
            D[0][i]=2*(i+1)*pow(delta,3);
        }
        clock_t begin,end;
        begin=clock();
        //so need to set up different grid dimension for different value of j,
        //when j decrease the size of the thread using will decrease.
        dim3 dimGrid(4,1); //so we have 4 blocks each block will in charge a,b,c,d respectly.
        dim3 dimBlock(16,16);

        //m is the size
        float ** AT,**BT,**CT,**DT;
        int size=m*sizeof(float*);

        hipMalloc((void**)&AT,size);
        hipMalloc((void**)&BT,size);
        hipMalloc((void**)&CT,size);
        hipMalloc((void**)&DT,size);

        hipMemcpy(AT,A,size,hipMemcpyHostToDevice);
        hipMemcpy(BT,B,size,hipMemcpyHostToDevice);
        hipMemcpy(CT,C,size,hipMemcpyHostToDevice);
        hipMemcpy(DT,D,size,hipMemcpyHostToDevice);

        printf("this is to test EXPO should see 9 here: %d \n",EXPO);

        for(int j=1;j<EXPO;j++)
        {
            //for each j do the work sequentially, inside this loop do work parallel.
          int powerNumber=pow(2,j-1);
          int totalNumber=m+1;
           CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j,powerNumber,totalNumber,AT,BT,CT,DT);
           printf("called from host %d \n",j);
        }
        //copy data back to device
        hipMemcpy(A,AT,size,hipMemcpyDeviceToHost);
        hipMemcpy(B,BT,size,hipMemcpyDeviceToHost);
        hipMemcpy(C,CT,size,hipMemcpyDeviceToHost);
        hipMemcpy(D,DT,size,hipMemcpyDeviceToHost);
    
        double time_spent;


        end=clock();
        time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
        printf("time spend for 524 n points is :%f seconds \n",time_spent);

        for(int k=0;k<100;k++)
        {
         printf("A new 1: %f \n",A[1][k]);
          printf("A new 8: %f \n",A[8][k]);
        }
        
        hipFree(AT);
        hipFree(BT);
        hipFree(CT);
        hipFree(DT);
      //release memory
        for(int i=0;i<9;i++)
        {
            free(A[i]);
        }
        free(A);

        for(int i=0;i<9;i++)
        {
            free(B[i]);
        }
        free(B);

        for(int i=0;i<9;i++)
        {
            free(C[i]);
        }
        free(C);

        for(int i=0;i<9;i++)
        {
            free(D[i]);
        }
        free(D);

        return 0;
    }