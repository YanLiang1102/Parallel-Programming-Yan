#include <stdio.h>
#include <hip/hip_runtime.h>
 
const int N = 7; 
const int blocksize = 7; 
 
__global__ void hello(char *a, int *b) 
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;        // Finds the thread_id
        //a[threadIdx.x] += b[threadIdx.x];
       // a[idx] += b[idx];
       // printf("yan yan yan! \n");
      int blocksize=3;
      int gridsize=2;
      int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int blocktotal=blocksize*blocksize;
	int temp1=by*gridsize+bx;
	int temp2=ty*blocksize+tx;

	int ind=temp1*blocktotal+temp2;
	printf("%d \n",ind);

}
 
int main()
{
        char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0, -11, 1};
 
        char *ad;
        int *bd;
        const int csize = N*sizeof(char);
        const int isize = N*sizeof(int);
 
        //printf("The original string: %s\n", a);
 
        hipMalloc( (void**)&ad, csize ); 
        hipMalloc( (void**)&bd, isize ); 
        hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
        hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
        
        dim3 Block( 3, 3 );                     // Number of threads per block
        dim3 Grid( 2, 2 );              // Number of thread blocks
        
        hello<<<Grid, Block>>>(ad, bd);
        hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
        hipFree( ad );
        hipFree( bd );
       // printf("The modified string: %s\n", a);
               return 0;
}

