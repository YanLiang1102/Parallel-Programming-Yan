   #include <stdio.h>
 #include <hip/hip_runtime.h>
 #include <time.h>
 #define EXPO 3


__global__ void RecursiveDoublingKernel(int variableSize, int step,int blockRow, int blockColumn,float* deviceY,float* deviceM,int evenOrOddFlag,float deviceA,float* deviceB,float* deviceC, float *deviceD)
{
	//we weill do something like y(i+1)=my(i)+b
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processIndex=tx;
	printf("%d ",tx);

    printf("%f,%f,%f \n",deviceY[0],deviceY[1],deviceY[2]);
    printf("%f,%f,%f \n",deviceM[0],deviceM[1],deviceM[2]);
    

	//so M and Y will be divided into two part, the first part store the old value
	//the second half part store the updated value

	int halfSize=variableSize;
	//teh start index of the second part will be halfsize;
    //so if evenOrOddFlag is Odd, the new value will be stored in the second half,
    //otherwise it will be stored in the first half. 
    int secondhalfHelper=halfSize+step+processIndex;
    printf("second half helper is: %d \n",secondhalfHelper);

    //be careful that 1-step the old value still need to be copied to the current value,since the new value will start calculated at step+1

    if(evenOrOddFlag%2==1)
    {
    	printf("does this ever got run?");
      deviceY[secondhalfHelper]=deviceY[secondhalfHelper-halfSize]+deviceM[secondhalfHelper-halfSize]*deviceY[processIndex];
      deviceM[secondhalfHelper]=deviceM[secondhalfHelper-halfSize]*deviceM[processIndex];
      //copy it once here
      if(tx==0&&ty==0)
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i+halfSize]=deviceY[i];
          deviceM[i+halfSize]=deviceM[i];
      	}
      }
    }
    else
    {
       
       printf("this should not run \n");//so will store the new value in the first part
      deviceY[secondhalfHelper-halfSize]=deviceY[secondhalfHelper]+deviceM[secondhalfHelper]*deviceY[halfSize+processIndex];
      deviceM[secondhalfHelper-halfSize]=deviceM[secondhalfHelper]*deviceM[halfSize+processIndex];

       if(tx==0&&ty==0) //just need to copy once, so the other processors allow to idle at thsi time
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i]=deviceY[i+halfSize];
          deviceM[i+halfSize]=deviceM[i];
      	}
      }
    }
  


    __syncthreads();
}

__global__ void MatrixVersionRecursiveDoubling(int variableSize, int step,int blockRow, int blockColumn,float* deviceYForW,float* deviceMForW,int evenOrOddFlag,float* deviceA, float* deviceB, float* deviceC, float* deviceD)
{
  //so right now just use grid (1,1) if time allow will implment other grid size
  	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processId=tx; //this is only for the this particluar grid and block setup
    
    int halfSizeY=variableSize;
    int halfSizeM=2*variableSize;
/*
    int secondhalfHelper=halfSize+step+2*processIndex; //this need to multiply 2, different from non-matrix version
    int secondhalfHelper1=halfSize+step+4*processIndex;*/

    int indexHelperY=halfSizeY+2*step+2*processId;
    int indexHelperM=halfSizeM+4*step+4*processId;

    if(evenOrOddFlag%2==1)
    {
    	//update M and Y here
    	deviceYForW[indexHelperY]=deviceYForW[indexHelperY-halfSizeY]+deviceMForW[indexHelperM-halfSizeM]*deviceYForW[2*processId]+deviceMForW[indexHelperM-halfSizeM+1]*deviceYForW[2*processId+1];
    	deviceYForW[indexHelperY+1]=deviceYForW[indexHelperY-halfSizeY+1]+deviceMForW[indexHelperM-halfSizeM+2]*deviceYForW[2*processId]+deviceMForW[indexHelperM-halfSizeM+3]*deviceYForW[2*processId+1];

        deviceMForW[indexHelperM]=deviceMForW[4*step+4*processId]*deviceMForW[4*processId]+deviceMForW[4*step+4*processId+1]*deviceMForW[4*processId+2];
        deviceMForW[indexHelperM+1]=deviceMForW[4*step+4*processId]*deviceMForW[4*processId+1]+deviceMForW[4*step+4*processId+1]*deviceMForW[4*processId+3];
        deviceMForW[indexHelperM+2]=deviceMForW[4*step+4*processId+2]*deviceMForW[4*processId]+deviceMForW[4*step+4*processId+3]*deviceMForW[4*processId+2];
        deviceMForW[indexHelperM+3]=deviceMForW[4*step+4*processId+2]*deviceMForW[4*processId+1]+deviceMForW[4*step+4*processId+3]*deviceMForW[4*processId+3];

        //now need to copy 1-- step old value to new value just need to copy once for each step
        for(int i=0;i<step;i++)
        {
        	deviceYForW[halfSizeY+2*i]=deviceYForW[2*i];
        	deviceYForW[halfSizeY+2*i+1]=deviceYForW[2*i+1];

        	deviceMForW[halfSizeM+4*i]=deviceMForW[4*i];
        	deviceMForW[halfSizeM+4*i+1]=deviceMForW[4*i+1];
        	deviceMForW[halfSizeM+4*i+2]=deviceMForW[4*i+2];
        	deviceMForW[halfSizeM+4*i+3]=deviceMForW[4*i+3];
        }
    }
    else
    {
        deviceYForW[indexHelperY-halfSizeY]=deviceYForW[indexHelperY]+deviceMForW[indexHelperM]*deviceYForW[2*processId+halfSizeY]+deviceMForW[indexHelperM+1]*deviceYForW[2*processId+1+halfSizeY];
    	deviceYForW[indexHelperY-halfSizeY+1]=deviceYForW[indexHelperY+1]+deviceMForW[indexHelperM+2]*deviceYForW[2*processId+halfSizeY]+deviceMForW[indexHelperM+3]*deviceYForW[2*processId+1+halfSizeY];

        deviceMForW[indexHelperM-halfSizeM]=deviceMForW[4*step+4*processId+halfSizeM]*deviceMForW[4*processId+halfSizeM]+deviceMForW[4*step+4*processId+1+halfSizeM]*deviceMForW[4*processId+2+halfSizeM];
        deviceMForW[indexHelperM+1-halfSizeM]=deviceMForW[4*step+4*processId+halfSizeM]*deviceMForW[4*processId+1+halfSizeM]+deviceMForW[4*step+4*processId+1+halfSizeM]*deviceMForW[4*processId+3+halfSizeM];
        deviceMForW[indexHelperM+2-halfSizeM]=deviceMForW[4*step+4*processId+2+halfSizeM]*deviceMForW[4*processId+halfSizeM]+deviceMForW[4*step+4*processId+3+halfSizeM]*deviceMForW[4*processId+2+halfSizeM];
        deviceMForW[indexHelperM+3-halfSizeM]=deviceMForW[4*step+4*processId+2+halfSizeM]*deviceMForW[4*processId+1+halfSizeM]+deviceMForW[4*step+4*processId+3+halfSizeM]*deviceMForW[4*processId+3+halfSizeM];

        //now need to copy 1-- step old value to new value just need to copy once for each step
        for(int i=0;i<step;i++)
        {
        	deviceYForW[2*i]=deviceYForW[2*i+halfSizeY];
        	deviceYForW[2*i+1]=deviceYForW[2*i+1+halfSizeY];

        	deviceMForW[4*i]=deviceMForW[4*i+halfSizeM];
        	deviceMForW[4*i+1]=deviceMForW[4*i+1+halfSizeM];
        	deviceMForW[4*i+2]=deviceMForW[4*i+2+halfSizeM];
        	deviceMForW[4*i+3]=deviceMForW[4*i+3+halfSizeM];
        }

    }


}

int main()
{
/* float* M;
 float* Y;
 int variableSize=10;
 int variableSpace=2*variableSize*sizeof(float);*/
   //make it double size since it run in parallel so you want to keep all the previous version
/* M=(float*)malloc(variableSpace);
 Y=(float*)malloc(variableSpace); */

/* M[0]=1;
 Y[0]=1;*/

 int m=pow(2,EXPO)-1; 
 int b=1;
 int a=0;
 float delta=(b-a)*1.0/(m+1.0);

//store teh metrix that is to be LU decomposited
 float *A;
 float *B;
 float *C;
 float *D;


	int chunkLength=m;
	int chunkSize=chunkLength*sizeof(float);
	A=(float*)malloc(chunkSize);
	B=(float*)malloc(chunkSize);
	C=(float*)malloc(chunkSize);
	D=(float*)malloc(chunkSize);

	 A[0]=0;
        //int vectorLength=EXPO*m;
     
        for(int i=1;i<m;i++)
        {
            A[i]=1-delta*delta*0.5*(i+1);
        }
        //else will be 0
      /*  for(int i=m;i<chunkLength;i++)
        {
          A[i]=0;
        }*/

        for(int i=0;i<m;i++)
        {
            B[i]=-2+delta*delta*1.0;
        }
      /*    for(int i=m;i<chunkLength;i++)
        {
          B[i]=0;
        }*/

        C[m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[i]=1+0.5*delta*delta*(i+1);
        }
     /*   for(int i=m;i<chunkLength;i++)
        {
          C[i]=0;
        }*/


       /* D[0]=2*delta*delta*delta+0.5*delta*delta-1;*/
        for(int i=0;i<m-1;i++)
        {
            D[i]=2*(i+1)*pow(delta,3);
        }
        D[m-1]=2*m*delta*delta*delta-1+3.5*delta*delta;
      /*  for(int i=m;i<chunkLength;i++)
        {
          D[i]=0;
        }*/

       float *deviceA, *deviceB, *deviceC, *deviceD;
        hipMalloc((void**)&deviceA,chunkSize);
        hipMalloc((void**)&deviceB,chunkSize);
        hipMalloc((void**)&deviceC,chunkSize);
        hipMalloc((void**)&deviceD,chunkSize);     
       //copy the host vector to device.
        hipMemcpy(deviceA,A,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceD,D,chunkSize,hipMemcpyHostToDevice);

        clock_t begin,end;
        begin=clock();

     //start the code to calculate the w with recursive doubling applied to matrix
      //so we need 2*2*(N-1) for both YforW and 2*4*(N-1) for MforW , the size N should be equal to m here
     float *MforW, *YforW;
     

     int MforWLength=4*(m-1);
     int YforWLength=2*(m-1);
     int MforWSize=2*MforWLength*sizeof(float);
     int YforWSize=2*YforWLength*sizeof(float);
     MforW=(float*)malloc(MforWSize);
     YforW=(float*)malloc(YforWSize);

     //the first step of recursive doubling, initialize Y and M;
     YforW[0]=1;
     YforW[1]=B[0]/(C[0]*1.0);
     //the other should be 0 since V(I)=A[I]V[I-1]+0
     for(int i=2;i<YforWLength;i++)
     {
     	YforW[i]=0;
     }
     //the first one for M should be[1,0,0,1]
     MforW[0]=1;
     MforW[1]=0;
     MforW[2]=0;
     MforW[3]=1;
     for(int i=4;i<MforWLength;i=i+4)
     {
     	MforW[i]=0;
     	MforW[i+1]=1;
     	MforW[i+2]=-1.0*A[i/4]/C[i/4];
     	MforW[i+3]=1.0*B[i/4]/C[i/4];
     }

     float *deviceMforW, *deviceYforW;
     hipMalloc((void**)&deviceMforW,MforWSize);
     hipMalloc((void**)&deviceYforW,YforWSize);

     hipMemcpy(deviceMforW,MforW,MforWSize,hipMemcpyHostToDevice);
     hipMemcpy(deviceYforW,YforW,YforWSize,hipMemcpyHostToDevice);

   
   int step=1;
   int evenOrOddFlag=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlag=evenOrOddFlag+1;
  	  dim3 dimGrid(1,1);
  	  int blockRow=1;
  	  int blockColumn=(m-1)-step;
  	  dim3 dimBlock(blockColumn,blockRow);
  	  //variableSIZE should be half size y
  	  MatrixVersionRecursiveDoubling<<<dimGrid,dimBlock>>>(YforWLength,step,blockRow,blockColumn,deviceYforW,deviceMforW,evenOrOddFlag,deviceA,deviceB,deviceC,deviceD);
        step=step+step;
    
   }while( step <= YforWLength/2);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   hipMemcpy(MforW,deviceMforW,MforWSize,hipMemcpyDeviceToHost);
   hipMemcpy(YforW,deviceYforW,YforWSize,hipMemcpyDeviceToHost);


/*M[0]=1;
Y[0]=1;

 for(int i=1;i<variableSize;i++)
 {
 	M[i]=2;
 	Y[i]=3;
 }
 float *deviceM, *deviceY;
 cudaMalloc((void**)&deviceM,variableSpace);
 cudaMalloc((void**)&deviceY,variableSpace);

 cudaMemcpy(deviceM,M,variableSpace,cudaMemcpyHostToDevice);
 cudaMemcpy(deviceY,Y,variableSpace,cudaMemcpyHostToDevice);

   
   int step=1;
   int evenOrOddFlag=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlag=evenOrOddFlag+1;
  	  dim3 dimGrid(1,1);
  	  int blockRow=1;
  	  int blockColumn=variableSize-step;
  	  dim3 dimBlock(blockColumn,blockRow);
  	  RecursiveDoublingKernel<<<dimGrid,dimBlock>>>(variableSize,step,blockRow,blockColumn,deviceY,deviceM,evenOrOddFlag);
        step=step+step;
      
    
   }while( step <= variableSize);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   cudaMemcpy(M,deviceM,variableSpace,cudaMemcpyDeviceToHost);
   cudaMemcpy(Y,deviceY,variableSpace,cudaMemcpyDeviceToHost);*/
/*   printf("solution is here: \n");
   if(evenOrOddFlag%2==0)
   {
     for(int i=0;i<variableSize;i++)
     {
     	printf("%f \n",Y[i]);
     }
   }
   else
   {
   	  for(int i=0;i<variableSize;i++)
     {
     	printf("%f \n",Y[i+variableSize]);
     }
   }*/

      printf("solution is here: \n");
   if(evenOrOddFlag%2==0)
   {
     for(int i=0;i<YforWLength;i++)
     {
     	printf("%f \n",YforW[i]);
     }
   }
   else
   {
   	  for(int i=0;i<YforWLength;i++)
     {
     	printf("%f \n",YforW[i+YforWLength]);
     }
   }
      double time_spent;
      end=clock();
      time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
      printf("\n time used to calculate this is :%f seconds \n",time_spent);
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        hipFree(deviceD);
        hipFree(deviceMforW);
        hipFree(deviceYforW);
       
        free(A);
        free(B);
        free(C);
        free(D);
        free(MforW);
        free(YforW);

  return 0;
}

