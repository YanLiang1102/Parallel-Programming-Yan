   #include <stdio.h>
 #include <hip/hip_runtime.h>
 #include <time.h>
 #define EXPO 7


__global__ void RecursiveDoublingKernel(int variableSize, int step,int blockRow, int blockColumn,float* deviceY,float* deviceM,int evenOrOddFlag)
{
	//we weill do something like y(i+1)=my(i)+b
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processIndex=tx;
/*	printf("%d ",tx);

    printf("%f,%f,%f \n",deviceY[0],deviceY[1],deviceY[2]);
    printf("%f,%f,%f \n",deviceM[0],deviceM[1],deviceM[2]);*/
    

	//so M and Y will be divided into two part, the first part store the old value
	//the second half part store the updated value

	int halfSize=variableSize;
	

	//teh start index of the second part will be halfsize;
    //so if evenOrOddFlag is Odd, the new value will be stored in the second half,
    //otherwise it will be stored in the first half. 
    int secondhalfHelper=halfSize+step+processIndex;
    //printf("second half helper is: %d \n",secondhalfHelper);

    //be careful that 1-step the old value still need to be copied to the current value,since the new value will start calculated at step+1

    if(evenOrOddFlag%2==1)
    {
    	//printf("does this ever got run?");
      deviceY[secondhalfHelper]=deviceY[secondhalfHelper-halfSize]+deviceM[secondhalfHelper-halfSize]*deviceY[processIndex];
      deviceM[secondhalfHelper]=deviceM[secondhalfHelper-halfSize]*deviceM[processIndex];
      //copy it once here
      if(tx==0&&ty==0)
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i+halfSize]=deviceY[i];
          deviceM[i+halfSize]=deviceM[i];
      	}
      }
    }
    else
    {
       
      deviceY[secondhalfHelper-halfSize]=deviceY[secondhalfHelper]+deviceM[secondhalfHelper]*deviceY[halfSize+processIndex];
      deviceM[secondhalfHelper-halfSize]=deviceM[secondhalfHelper]*deviceM[halfSize+processIndex];

       if(tx==0&&ty==0) //just need to copy once, so the other processors allow to idle at thsi time
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i]=deviceY[i+halfSize];
         deviceM[i]=deviceM[i+halfSize];
      	}
      }
    }
    __syncthreads();
}

__global__ void LoopingbackRecursiveDoublingKernel(int variableSize, int step,int blockRow, int blockColumn,float* deviceY,float* deviceM,int evenOrOddFlag)
{
	//we weill do something like y(i+1)=my(i)+b
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processIndex=tx;
/*	printf("%d ",tx);

    printf("%f,%f,%f \n",deviceY[0],deviceY[1],deviceY[2]);
    printf("%f,%f,%f \n",deviceM[0],deviceM[1],deviceM[2]);*/
    

	//so M and Y will be divided into two part, the first part store the old value
	//the second half part store the updated value

	int halfSize=variableSize;
	

	//teh start index of the second part will be halfsize;
    //so if evenOrOddFlag is Odd, the new value will be stored in the second half,
    //otherwise it will be stored in the first half. 
    //int secondhalfHelper=halfSize+step+processIndex;
    int secondhalfHelper=halfSize+processIndex;

    //printf("second half helper is: %d \n",secondhalfHelper);

    //be careful that 1-step the old value still need to be copied to the current value,since the new value will start calculated at step+1

    if(evenOrOddFlag%2==1)
    {
      deviceY[secondhalfHelper]=deviceY[processIndex]+deviceY[processIndex+step]*deviceM[processIndex];
      deviceM[secondhalfHelper]=deviceM[processIndex]*deviceM[processIndex+step];

      //now the reverse part need to copy the second part
      //should be from index N-i to index variableSize-1
      if(tx==0&&ty==0)
      {
      	for(int i=variableSize-step;i<variableSize;i++)
      	{
          deviceY[i+halfSize]=deviceY[i];
          deviceM[i+halfSize]=deviceM[i];
      	}
      }
    }
    else
    {
       
      deviceY[processIndex]=deviceY[halfSize+processIndex]+deviceY[halfSize+step+processIndex]*deviceM[halfSize+processIndex]; 
      //deviceY[secondhalfHelper-halfSize]=deviceY[secondhalfHelper]+deviceM[secondhalfHelper]*deviceY[halfSize+processIndex];
      //deviceM[secondhalfHelper-halfSize]=deviceM[secondhalfHelper]*deviceM[halfSize+processIndex];
      deviceM[processIndex]=deviceM[processIndex+halfSize]*deviceM[processIndex+halfSize+step];
      if(tx==0&&ty==0)
      {
      	for(int i=variableSize-step;i<variableSize;i++)
      	{
          deviceY[i]=deviceY[i+halfSize];
          deviceM[i]=deviceM[i+halfSize];
      	}
      }
    }
    __syncthreads();
}


__global__ void MatrixVersionRecursiveDoubling(int variableSize, int step,int blockRow, int blockColumn,float* deviceYForW,float* deviceMForW,int evenOrOddFlag,float* deviceA, float* deviceB, float* deviceC, float* deviceD)
{
  //so right now just use grid (1,1) if time allow will implment other grid size
  	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processId=tx; //this is only for the this particluar grid and block setup
    
    int halfSizeY=variableSize;
    int halfSizeM=2*variableSize;
/*
    int secondhalfHelper=halfSize+step+2*processIndex; //this need to multiply 2, different from non-matrix version
    int secondhalfHelper1=halfSize+step+4*processIndex;*/

    int indexHelperY=halfSizeY+2*step+2*processId;
    int indexHelperM=halfSizeM+4*step+4*processId;

    if(evenOrOddFlag%2==1)
    {
    	//update M and Y here
    	deviceYForW[indexHelperY]=deviceYForW[indexHelperY-halfSizeY]+deviceMForW[indexHelperM-halfSizeM]*deviceYForW[2*processId]+deviceMForW[indexHelperM-halfSizeM+1]*deviceYForW[2*processId+1];
    	deviceYForW[indexHelperY+1]=deviceYForW[indexHelperY-halfSizeY+1]+deviceMForW[indexHelperM-halfSizeM+2]*deviceYForW[2*processId]+deviceMForW[indexHelperM-halfSizeM+3]*deviceYForW[2*processId+1];

        deviceMForW[indexHelperM]=deviceMForW[4*step+4*processId]*deviceMForW[4*processId]+deviceMForW[4*step+4*processId+1]*deviceMForW[4*processId+2];
        deviceMForW[indexHelperM+1]=deviceMForW[4*step+4*processId]*deviceMForW[4*processId+1]+deviceMForW[4*step+4*processId+1]*deviceMForW[4*processId+3];
        deviceMForW[indexHelperM+2]=deviceMForW[4*step+4*processId+2]*deviceMForW[4*processId]+deviceMForW[4*step+4*processId+3]*deviceMForW[4*processId+2];
        deviceMForW[indexHelperM+3]=deviceMForW[4*step+4*processId+2]*deviceMForW[4*processId+1]+deviceMForW[4*step+4*processId+3]*deviceMForW[4*processId+3];

        //now need to copy 1-- step old value to new value just need to copy once for each step
        for(int i=0;i<step;i++)
        {
        	deviceYForW[halfSizeY+2*i]=deviceYForW[2*i];
        	deviceYForW[halfSizeY+2*i+1]=deviceYForW[2*i+1];

        	deviceMForW[halfSizeM+4*i]=deviceMForW[4*i];
        	deviceMForW[halfSizeM+4*i+1]=deviceMForW[4*i+1];
        	deviceMForW[halfSizeM+4*i+2]=deviceMForW[4*i+2];
        	deviceMForW[halfSizeM+4*i+3]=deviceMForW[4*i+3];
        }
    }
    else
    {
        deviceYForW[indexHelperY-halfSizeY]=deviceYForW[indexHelperY]+deviceMForW[indexHelperM]*deviceYForW[2*processId+halfSizeY]+deviceMForW[indexHelperM+1]*deviceYForW[2*processId+1+halfSizeY];
    	deviceYForW[indexHelperY-halfSizeY+1]=deviceYForW[indexHelperY+1]+deviceMForW[indexHelperM+2]*deviceYForW[2*processId+halfSizeY]+deviceMForW[indexHelperM+3]*deviceYForW[2*processId+1+halfSizeY];

        deviceMForW[indexHelperM-halfSizeM]=deviceMForW[4*step+4*processId+halfSizeM]*deviceMForW[4*processId+halfSizeM]+deviceMForW[4*step+4*processId+1+halfSizeM]*deviceMForW[4*processId+2+halfSizeM];
        deviceMForW[indexHelperM+1-halfSizeM]=deviceMForW[4*step+4*processId+halfSizeM]*deviceMForW[4*processId+1+halfSizeM]+deviceMForW[4*step+4*processId+1+halfSizeM]*deviceMForW[4*processId+3+halfSizeM];
        deviceMForW[indexHelperM+2-halfSizeM]=deviceMForW[4*step+4*processId+2+halfSizeM]*deviceMForW[4*processId+halfSizeM]+deviceMForW[4*step+4*processId+3+halfSizeM]*deviceMForW[4*processId+2+halfSizeM];
        deviceMForW[indexHelperM+3-halfSizeM]=deviceMForW[4*step+4*processId+2+halfSizeM]*deviceMForW[4*processId+1+halfSizeM]+deviceMForW[4*step+4*processId+3+halfSizeM]*deviceMForW[4*processId+3+halfSizeM];

        //now need to copy 1-- step old value to new value just need to copy once for each step
        for(int i=0;i<step;i++)
        {
        	deviceYForW[2*i]=deviceYForW[2*i+halfSizeY];
        	deviceYForW[2*i+1]=deviceYForW[2*i+1+halfSizeY];

        	deviceMForW[4*i]=deviceMForW[4*i+halfSizeM];
        	deviceMForW[4*i+1]=deviceMForW[4*i+1+halfSizeM];
        	deviceMForW[4*i+2]=deviceMForW[4*i+2+halfSizeM];
        	deviceMForW[4*i+3]=deviceMForW[4*i+3+halfSizeM];
        }

    }
    __syncthreads();


}

int main()
{
/* float* M;
 float* Y;
 int variableSize=10;
 int variableSpace=2*variableSize*sizeof(float);*/
   //make it double size since it run in parallel so you want to keep all the previous version
/* M=(float*)malloc(variableSpace);
 Y=(float*)malloc(variableSpace); */

/* M[0]=1;
 Y[0]=1;*/

 int m=pow(2,EXPO)-1; 
 int b=1;
 int a=0;
 float delta=(b-a)*1.0/(m+1.0);

//store teh metrix that is to be LU decomposited
 float *A;
 float *B;
 float *C;
 float *D;
 float *W;
 float *G;

	int chunkLength=m;
	int chunkSize=chunkLength*sizeof(float);
	A=(float*)malloc(chunkSize);
	B=(float*)malloc(chunkSize);
	C=(float*)malloc(chunkSize);
	D=(float*)malloc(chunkSize);
	W=(float*)malloc((m-1)*sizeof(float));
	G=(float*)malloc((m*sizeof(float)));

	 A[0]=0;
        //int vectorLength=EXPO*m;
     
        for(int i=1;i<m;i++)
        {
            A[i]=1-delta*delta*0.5*(i+1);
        }
        //else will be 0
      /*  for(int i=m;i<chunkLength;i++)
        {
          A[i]=0;
        }*/

        for(int i=0;i<m;i++)
        {
            B[i]=-2+delta*delta*1.0;
        }
      /*    for(int i=m;i<chunkLength;i++)
        {
          B[i]=0;
        }*/

        C[m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            C[i]=1+0.5*delta*delta*(i+1);
        }
     /*   for(int i=m;i<chunkLength;i++)
        {
          C[i]=0;
        }*/


        
        for(int i=0;i<m-1;i++)
        {
            D[i]=2*(i+1)*pow(delta,3);
        }
        D[m-1]=2*m*delta*delta*delta-1+3.5*delta*delta;
      /*  for(int i=m;i<chunkLength;i++)
        {
          D[i]=0;
        }*/

       float *deviceA, *deviceB, *deviceC, *deviceD;
        hipMalloc((void**)&deviceA,chunkSize);
        hipMalloc((void**)&deviceB,chunkSize);
        hipMalloc((void**)&deviceC,chunkSize);
        hipMalloc((void**)&deviceD,chunkSize);     
       //copy the host vector to device.
        hipMemcpy(deviceA,A,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceD,D,chunkSize,hipMemcpyHostToDevice);

        clock_t begin,end;
        begin=clock();

     //start the code to calculate the w with recursive doubling applied to matrix
      //so we need 2*2*(N-1) for both YforW and 2*4*(N-1) for MforW , the size N should be equal to m here
     float *MforW, *YforW;
     

     int MforWLength=4*(m-1);
     int YforWLength=2*(m-1);
     int MforWSize=2*MforWLength*sizeof(float);
     int YforWSize=2*YforWLength*sizeof(float);
     MforW=(float*)malloc(MforWSize);
     YforW=(float*)malloc(YforWSize);

     //the first step of recursive doubling, initialize Y and M;
     YforW[0]=1;
     YforW[1]=B[0]/(C[0]*1.0);
     //the other should be 0 since V(I)=A[I]V[I-1]+0
     for(int i=2;i<YforWLength;i++)
     {
     	YforW[i]=0;
     }
     //the first one for M should be[1,0,0,1]
     MforW[0]=1;
     MforW[1]=0;
     MforW[2]=0;
     MforW[3]=1;
     for(int i=4;i<MforWLength;i=i+4)
     {
     	MforW[i]=0;
     	MforW[i+1]=1;
     	MforW[i+2]=-1.0*A[i/4]/C[i/4];
     	MforW[i+3]=1.0*B[i/4]/C[i/4];
     }

     float *deviceMforW, *deviceYforW;
     hipMalloc((void**)&deviceMforW,MforWSize);
     hipMalloc((void**)&deviceYforW,YforWSize);

     hipMemcpy(deviceMforW,MforW,MforWSize,hipMemcpyHostToDevice);
     hipMemcpy(deviceYforW,YforW,YforWSize,hipMemcpyHostToDevice);

   
   int step=1;
   int evenOrOddFlag=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlag=evenOrOddFlag+1;
  	  dim3 dimGrid(1,1);
  	  int blockRow=1;
  	  int blockColumn=(m-1)-step;
  	  dim3 dimBlock(blockColumn,blockRow);
  	  //variableSIZE should be half size y
  	  MatrixVersionRecursiveDoubling<<<dimGrid,dimBlock>>>(YforWLength,step,blockRow,blockColumn,deviceYforW,deviceMforW,evenOrOddFlag,deviceA,deviceB,deviceC,deviceD);
        step=step+step;
    
   }while( step <= YforWLength/2);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   hipMemcpy(MforW,deviceMforW,MforWSize,hipMemcpyDeviceToHost);
   hipMemcpy(YforW,deviceYforW,YforWSize,hipMemcpyDeviceToHost);

      printf("The following are w value from recursvie doubling: \n");
   if(evenOrOddFlag%2==0)
   {
   	//length of w is m-1 and length of y is s(m-1)
     for(int i=0;i<m-1;i++)
     {
     	if(i%16==0)
     	{
     		printf("\n");
     	}
     	W[i]=YforW[2*i]*1.0/YforW[2*i+1];
     	printf("%f ",W[i]);
     }
   }
   else
   {
   	   for(int i=0;i<m-1;i++)
     {
     	if(i%16==0)
     	{
     		printf("\n");
     	}
     	W[i]=YforW[2*i+YforWLength]*1.0/YforW[2*i+1+YforWLength];
     	printf("%f ",W[i]);
     }
   }  

   //now we get the w value, next step is to get the g value
   //g will have n-1 in values.
   //according to the formula 5.3.3.7
   float* MforG,*YforG;
   MforG=(float*)malloc(m*sizeof(float));
   YforG=(float*)malloc(m*sizeof(float));
   int forGSize=2*m*sizeof(float);
   YforG[0]=D[0]*1.0/B[0];
   MforG[0]=1.0;
/*  printf("\n test start here");*/
	 for(int i=1;i<m;i++)
	 {
	 	YforG[i]=D[i]/(B[i]-A[i]*W[i-1]);
	 	MforG[i]=-1*A[i]/(B[i]-A[i]*W[i-1]);
	 }

	 

	 float *deviceMforG, *deviceYforG;
	 hipMalloc((void**)&deviceMforG,forGSize);
	 hipMalloc((void**)&deviceYforG,forGSize);


 hipMemcpy(deviceMforG,MforG,forGSize,hipMemcpyHostToDevice);
 hipMemcpy(deviceYforG,YforG,forGSize,hipMemcpyHostToDevice);

  int stepG=1;
  int evenOrOddFlagG=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlagG=evenOrOddFlagG+1;
  	  dim3 dimGrid1(1,1);
  	  int blockRow1=1;
  	  int blockColumn1=m-stepG;
  	  dim3 dimBlock1(blockColumn1,blockRow1);
  	  RecursiveDoublingKernel<<<dimGrid1,dimBlock1>>>(m,stepG,blockRow1,blockColumn1,deviceYforG,deviceMforG,evenOrOddFlagG);
      stepG=stepG+stepG;
   }while( stepG <= m);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   hipMemcpy(MforG,deviceMforG,forGSize,hipMemcpyDeviceToHost);
   hipMemcpy(YforG,deviceYforG,forGSize,hipMemcpyDeviceToHost);

   if(evenOrOddFlagG%2==0)
   {
   	
     for(int i=0;i<m;i++)
     {
		     	if(i%16==0)
		   	{
		   		printf("\n");
		   	}
		 G[i]=YforG[i];
     	printf("[%d] %f ",i,YforG[i]);
     }
   }
   else
   {

   	  for(int i=0;i<m;i++)
     {
	     	if(i%16==0)
	   	{
	   		printf("\n");
	   	}
	   	 G[i]=YforG[i];
     	printf("[%d] %f ",i,YforG[i+m]);
     }
   }


   //now we get G, it is time for us to reverse it back to get our final x
  float* MforX,*YforX;
   MforX=(float*)malloc(m*sizeof(float));
   YforX=(float*)malloc(m*sizeof(float));
   int forXSize=2*m*sizeof(float);
   YforX[m-1]=G[m-1];
   MforG[m-1]=1.0;
/*  printf("\n test start here");*/
	 for(int i=0;i<m-1;i++)
	 {
	 	YforX[i]=G[i];
	 	MforX[i]=-1*W[i];
	 }

	 

	 float *deviceMforX, *deviceYforX;
	 hipMalloc((void**)&deviceMforX,forXSize);
	 hipMalloc((void**)&deviceYforX,forXSize);


 hipMemcpy(deviceMforX,MforX,forXSize,hipMemcpyHostToDevice);
 hipMemcpy(deviceYforX,YforX,forXSize,hipMemcpyHostToDevice);

  int stepX=1;
  int evenOrOddFlagX=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlagX=evenOrOddFlagX+1;
  	  dim3 dimGrid2(1,1);
  	  int blockRow2=1;
  	  int blockColumn2=m-stepX;
  	  dim3 dimBlock2(blockColumn2,blockRow2);
  	  LoopingbackRecursiveDoublingKernel<<<dimGrid2,dimBlock2>>>(m,stepX,blockRow2,blockColumn2,deviceYforX,deviceMforX,evenOrOddFlagX);
      stepX=stepX+stepX;
   }while( stepX<= m);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   hipMemcpy(MforX,deviceMforX,forXSize,hipMemcpyDeviceToHost);
   hipMemcpy(YforX,deviceYforX,forXSize,hipMemcpyDeviceToHost);

   printf("The following is the result for x finally! \n");
   if(evenOrOddFlagX%2==0)
   {
   	
     for(int i=0;i<m;i++)
     {
		     	if(i%16==0)
		   	{
		   		printf("\n");
		   	}
     	printf(" %f ",YforX[i]);
     }
   }
   else
   {

   	  for(int i=0;i<m;i++)
     {
	     	if(i%16==0)
	   	{
	   		printf("\n");
	   	}
     	printf("%f ",YforX[i+m]);
     }
   }



     //printf("y for G is %f \n",YforG[444]);
      double time_spent;
      end=clock();
      time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
      printf("\n time used to calculate pde with %d varaible recursive doubling is :%f seconds \n",m,time_spent);

      hipFree(deviceA);
      hipFree(deviceB);
      hipFree(deviceC);
      hipFree(deviceD);
      hipFree(deviceMforW);
      hipFree(deviceYforW);
       
        free(A);
        free(B);
        free(C);
        free(D);
        free(MforW);
        free(YforW);

  return 0;
}

