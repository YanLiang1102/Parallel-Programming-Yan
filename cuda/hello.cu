#include <stdio.h>
#include <hip/hip_runtime.h>
 
const int N = 7; 
const int blocksize = 7; 
 
__global__ void hello(char *a, int *b) 
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;        // Finds the thread_id
        //a[threadIdx.x] += b[threadIdx.x];
        a[idx] += b[idx];
        printf("yan yan yan! \n");
}
 
int main()
{
        char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0, -11, 1};
 
        char *ad;
        int *bd;
        const int csize = N*sizeof(char);
        const int isize = N*sizeof(int);
 
        printf("The original string: %s\n", a);
 
        hipMalloc( (void**)&ad, csize ); 
        hipMalloc( (void**)&bd, isize ); 
        hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
        hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
        
        dim3 Block( 1, 1 );                     // Number of threads per block
        dim3 Grid( blocksize, 1 );              // Number of thread blocks
        
        hello<<<Grid, Block>>>(ad, bd);
        hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
        hipFree( ad );
        hipFree( bd );
        printf("The modified string: %s\n", a);
               return 0;
}

