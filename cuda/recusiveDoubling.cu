   #include <stdio.h>
 #include <hip/hip_runtime.h>
 #include <time.h>
 #define VARCOUNT 3


__global__ void RecursiveDoublingKernel(int variableSize, int step,int blcokRow, int blockColumn,float* deviceY,float* deviceM,int evenOrOddFlag)
{
	//we weill do something like y(i+1)=my(i)+b
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int processIndex=tx;
	printf("%d ",tx);

    printf("%f,%f,%f \n",deviceY[0],deviceY[1],deviceY[2]);
    printf("%f,%f,%f \n",deviceM[0],deviceM[1],deviceM[2]);
    

	//so M and Y will be divided into two part, the first part store the old value
	//the second half part store the updated value

	int halfSize=variableSize;
	//teh start index of the second part will be halfsize;
    //so if evenOrOddFlag is Odd, the new value will be stored in the second half,
    //otherwise it will be stored in the first half. 
    int secondhalfHelper=halfSize+step+processIndex;
    printf("second half helper is: %d \n",secondhalfHelper);

    //be careful that 1-step the old value still need to be copied to the current value,since the new value will start calculated at step+1

    if(evenOrOddFlag%2==1)
    {
    	printf("does this ever got run?");
      deviceY[secondhalfHelper]=deviceY[secondhalfHelper-halfSize]+deviceM[secondhalfHelper-halfSize]*deviceY[processIndex];
      deviceM[secondhalfHelper]=deviceM[secondhalfHelper-halfSize]*deviceM[processIndex];
      //copy it once here
      if(tx==0&&ty==0)
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i+halfSize]=deviceY[i];
          deviceM[i+halfSize]=deviceM[i];
      	}
      }
    }
    else
    {
       
       printf("this should not run \n");//so will store the new value in the first part
      deviceY[secondhalfHelper-halfSize]=deviceY[secondhalfHelper]+deviceM[secondhalfHelper]*deviceY[halfSize+processIndex];
      deviceM[secondhalfHelper-halfSize]=deviceM[secondhalfHelper]*deviceM[halfSize+processIndex];
      
       if(tx==0&&ty==0) //just need to copy once, so the other processors allow to idle at thsi time
      {
      	for(int i=0;i<step;i++)
      	{
          deviceY[i]=deviceY[i+halfSize];
          deviceM[i]=deviceM[i+halfSize];
      	}
      }
    }
  


    __syncthreads();
}

int main()
{
 float* M;
 float* Y;
 int variableSize=10;
 int variableSpace=2*variableSize*sizeof(float);
   //make it double size since it run in parallel so you want to keep all the previous version
 M=(float*)malloc(variableSpace);
 Y=(float*)malloc(variableSpace); 

 M[0]=1;
 Y[0]=1;

 for(int i=1;i<variableSize;i++)
 {
 	M[i]=2;
 	Y[i]=3;
 }
 float *deviceM, *deviceY;
 hipMalloc((void**)&deviceM,variableSpace);
 hipMalloc((void**)&deviceY,variableSpace);

 hipMemcpy(deviceM,M,variableSpace,hipMemcpyHostToDevice);
 hipMemcpy(deviceY,Y,variableSpace,hipMemcpyHostToDevice);

   
   int step=1;
   int evenOrOddFlag=0;

  do {
  	 //each time needs N-Step processors
  	
  	  evenOrOddFlag=evenOrOddFlag+1;
  	  dim3 dimGrid(1,1);
  	  int blockRow=1;
  	  int blockColumn=variableSize-step;
  	  dim3 dimBlock(blockColumn,blockRow);
  	  RecursiveDoublingKernel<<<dimGrid,dimBlock>>>(variableSize,step,blockRow,blockColumn,deviceY,deviceM,evenOrOddFlag);
        step=step+step;
      
    
   }while( step <= variableSize);

   //so if evenOrOddFlag is odd, it means that the latest value will be second half,
   //otherwise it will be in the first half
   hipMemcpy(M,deviceM,variableSpace,hipMemcpyDeviceToHost);
   hipMemcpy(Y,deviceY,variableSpace,hipMemcpyDeviceToHost);
   printf("solution is here: \n");
   if(evenOrOddFlag%2==0)
   {
     for(int i=0;i<variableSize;i++)
     {
     	printf("%f \n",Y[i]);
     }
   }
   else
   {
   	  for(int i=0;i<variableSize;i++)
     {
     	printf("%f \n",Y[i+variableSize]);
     }
   }
  /*   if(evenOrOddFlag%2==0)
   {
     for(int i=0;i<variableSize*2;i++)
     {
     	printf("%f \n",M[i]);
     }
   }
   else
   {
   	  for(int i=0;i<variableSize*2;i++)
     {
     	printf("%f \n",M[i+variableSize]);
     }
   }*/
  return 0;
}

