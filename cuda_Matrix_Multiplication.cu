#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
__global__ void MatrixMulKernel(float* Md, float* Nd,float* Pd,int Width)
{
	float Pvalue=0;

	for(int k=0;k<Width;++k)
	{
		float Melement=Md[threadIdx.y*Width+k];
		float Nelement=Nd[k*Width+threadIdx.x];
		Pvalue+=Melement*Nelement;
	}

	Pd[threadIdx.y*Width+threadIdx.x]=Pvalue;
}*/

	__global__ void MatrixMulKernel(float* M,float* N, float* Pd, int blockSize,int loopTimes)
	{
	__shared__ float Ms[1][1];
	__shared__ float Ns[1][1];
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int Row=by*blockSize+ty;
	int Col=bx*blockSize+tx;

	float sum=0;
	for(int m=0;m<loopTimes;m++)
	{
		Ms[ty][tx]=M[Row*blockSize*loopTimes+(m*blockSize+tx)];
		Ns[ty][tx]=N[Col+(m*blockSize+ty)*blockSize*loopTimes];
		__syncthreads();
		for(int j=0;j<blockSize;j++)
			sum+=Ms[ty][j]*Ns[j][tx];
		__syncthreads();
	}
    Pd[Row*blockSize*loopTimes+Col]=sum;

	}

int main()
{

	int matrixSize=2;//pow(2,8);
	int blockSize=1;//pow(2,4);
	//int noOfElement=matrixSize*matrixSize;
/*	float* M;
	float* N;
	float* P;
	M=malloc(4*sizeof(float));
	N=malloc(4*sizeof(float));
	P=malloc(4*sizeof(float));*/
	float M[matrixSize],N[matrixSize],P[matrixSize];

	for(int i=0;i<matrixSize*matrixSize;i++)
	{
		M[i]=i+1.0;
		N[i]=i+2.0;
		P[i]=0.0;
	}

	float* Pd;
    int size=matrixSize*matrixSize*sizeof(float);
	hipMalloc((void**)&Pd,size);
/*
	int size=Width*Width*sizeof(float);
    float* Md, *Nd, *Pd;

    hipMalloc((void**)&Md,size);
	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd,size);
	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd,size);

    dim3 dimGrid(1,1);
    dim3 dimBlock(Width,Width);
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	for(int i=0;i<4;i++)
	{
		printf("result: %f \n",P[i]);
	}*/
    dim3 dimGrid(matrixSize/blockSize,matrixSize/blockSize);
    dim3 dimBlock(blockSize,blockSize);
    MatrixMulKernel<<<dimGrid,dimBlock>>>(M,N,Pd,blockSize,matrixSize/blockSize);
    __syncthreads();

    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);


	hipFree(Pd);

	for(int i=0;i<matrixSize*matrixSize;i++)
	{
		printf("result: %f \n",P[i]);
	}


	return 0;
}