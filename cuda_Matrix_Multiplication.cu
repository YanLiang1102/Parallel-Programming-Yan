#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MatrixMulKernel(float* Md, float* Nd,float* Pd,int Width)
{
	float Pvalue=0;

	for(int k=0;k<Width;++k)
	{
		float Melement=Md[threadIdx.y*Width+k];
		float Nelement=Nd[k*Width+threadIdx.x];
		Pvalue+=Melement*Nelement;
	}

	Pd[threadIdx.y*Width+threadIdx.x]=Pvalue;
}

int main()
{
	int Width=2;
/*	float* M;
	float* N;
	float* P;
	M=malloc(4*sizeof(float));
	N=malloc(4*sizeof(float));
	P=malloc(4*sizeof(float));*/
	float M[2],N[2],P[2];

	for(int i=0;i<4;i++)
	{
		M[i]=i+1.0;
		N[i]=i+2.0;
		P[i]=0.0;
	}

	int size=Width*Width*sizeof(float);
    float* Md, *Nd, *Pd;

    hipMalloc((void**)&Md,size);
	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd,size);
	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd,size);

    dim3 dimGrid(1,1);
    dim3 dimBlock(Width,Width);
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	for(int i=0;i<4;i++)
	{
		printf("result: %f \n",P[i]);
	}


	return 0;
}