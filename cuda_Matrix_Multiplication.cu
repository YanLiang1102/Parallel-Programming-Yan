#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

    double r1();

	__global__ void MatrixMulKernel(float* M,float* N, float* Pd, int blockSize,int loopTimes)
	{
	__shared__ float Ms[16][16];
	__shared__ float Ns[16][16];
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;

	int Row=by*blockSize+ty;
	int Col=bx*blockSize+tx;

	float sum=0;
	for(int m=0;m<loopTimes;m++)
	{
		Ms[ty][tx]=M[Row*blockSize*loopTimes+(m*blockSize+tx)];
		Ns[ty][tx]=N[Col+(m*blockSize+ty)*blockSize*loopTimes];
		__syncthreads();
		for(int j=0;j<blockSize;j++)
			sum+=Ms[ty][j]*Ns[j][tx];
		__syncthreads();
	}
    Pd[Row*blockSize*loopTimes+Col]=sum;

	}

int main()
{

	int matrixSize=pow(2,8);
	int blockSize=pow(2,4); //the default blockSize I will put as is 16
	int noOfElement=matrixSize*matrixSize;
	float* M;
	float* N;
	float* P;
	M=(float*)malloc(noOfElement*sizeof(float));
	N=(float*)malloc(noOfElement*sizeof(float));
	P=(float*)malloc(noOfElement*sizeof(float));
	clock_t begin, end;


	for(int i=0;i<noOfElement;i++)
	{
		M[i]=r1();
		N[i]=r1();
		P[i]=0.0;
	}
	//start timing after generating the matrix
	begin = clock();

	float* Pd,*Md,*Nd;
    int size=noOfElement*sizeof(float);
	hipMalloc((void**)&Pd,size);

    //sned M and N to device
    hipMalloc((void**)&Md,size);
	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd,size);
	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
    dim3 dimGrid(matrixSize/blockSize,matrixSize/blockSize);
    dim3 dimBlock(blockSize,blockSize);
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,blockSize,matrixSize/blockSize);
 

    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);


	hipFree(Pd);

	
    double time_spent;

    
/* here, do your time-consuming job */
    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("time Spend for matrix size: (%d,%d), with blockSize: %d is :%f \n",matrixSize,matrixSize,blockSize,time_spent);
   

    printf("The following are the first 100 reuslt from the matrix multiplication:\n");
    //print out first 100 result.
	for(int i=0;i<100;i++)
	{
		printf("result: %f \n",P[i]);
	}

	return 0;
}

double r1()
{
    return -1.0*(double)rand() / (double)RAND_MAX ;
}