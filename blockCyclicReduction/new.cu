     #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #include <math.h>
    //#include <unistd.h>
    #define EXPO 2 //so [0,1] will be break into 2^6 intervals 64*64
    #define PI 3.14159265

    __global__ void CalculateTheD(int step,float* deviceB, float* deviceC, float* deviceD, float* deviceX, float* devicenewB, float* devicenewC)
    {
       int blocksize=16;
       int gridsize=gridDim.x; //this will always be 1
       // int gridsize2=gridDim.x;
       // printf("gridsize: %d %d\n", gridsize,gridsize2);
       // __device__ bool myGlobalFlag=true;
       
        int bx=blockIdx.x;
        int by=blockIdx.y;
        int tx=threadIdx.x;
        int ty=threadIdx.y;
        
        int blocktotal=blocksize*blocksize;
        int temp1=by*gridsize+bx;
        int temp2=ty*blocksize+tx;

       int ind=temp1*blocktotal+temp2;

      int m=(int)pow(2.0,EXPO*1.0)-1;
      /*int column=threadIdx.x;
      int row=threadIdx.y;*/
      int row=ind/m;
      int column=ind%m;

      int iloopstep=(int)pow(2.0,(EXPO-step)*1.0)-1;
      int h=(int)pow(2.0,(step-1)*1.0);
      int multiplier=(int)pow(2.0,step*1.0);
      int countHelper1=(int)pow(2.0,EXPO+1.0);
      int countHelper2=(int)pow(2.0,EXPO-step+2);

      float* oldB;
      float* oldC;
 

      float* newB;
      float* newC;
      //float* newD;
    
     //this is to exchange which hold the previous value which hold the current value
      if(step%2==0)
      {
      	oldB=devicenewB;
      	oldC=devicenewC;
      	/*oldD=devicenewD;*/

      	newB=deviceB;
      	newC=deviceC;
     /* 	newD=deviceD;*/
      }
      else
      {
      	oldB=deviceB;
      	oldC=deviceC;
      /*	oldD=deviceD;*/
        
        newB=devicenewB;
        newC=devicenewC;
  /*      newD=devicenewD;*/
      }

      //use the device value as old value and store the updated one in to the new value
      if(ind<m*m) //so only the first 63 threads do work and the other one is hanging there
      {
		    float sumBB=0.0;
		    for(int k=0;k<m;k++)
		    {
		      sumBB=sumBB+oldB[row*m+k]*oldB[k*m+column];
		    }
		    float sumCC=0.0;
		    for(int k=0;k<m;k++)
		    {
		      sumCC=sumCC+oldC[row*m+k]*oldC[k*m+column];
		    }

		    //based on formula (5.4.2.15) on book
		    newB[row*m+column]=2*sumCC-sumBB;
		    newC[row*m+column]=sumCC;

		    //now calculate the new d and it needs to loop through i in each block
		    //look at the third formula on 5.4.2.15 on book
		    ///D[ith BLOCK][j thSTep ]=D[(2^(k+1)-2^(k-j+1)+i-j-1)*m+...]
		    if(column==0)
		    {
		    	//for calculate d we just need 63 tthreads but B and C we need 63*63 threads
		    	//so in step :step, each thread will work on the row th value in each block in that step,sicne there are 63 threads.
		    	for(int i=1;i<=iloopstep;i++)
		    {

		    	float sumCD1=0.0;
		    	for(int k=0;k<m;k++)
		    	{

		    		sumCD1=sumCD1+oldC[row*m+k]*deviceD[(countHelper1-countHelper2+i*2-1-step)*m+k];
		    	}

		    	float sumCD2=0.0;
		    	for(int k=0;k<m;k++)
		    	{
		    		sumCD2=sumCD2+oldC[row*m+k]*deviceD[(countHelper1-countHelper2+i*2+1-step)*m+k];
		    	}

		    	float sumBD=0.0;
		    	for(int k=0;k<m;k++)
		    	{
		           sumBD=sumBD+oldB[row*m+k]*deviceD[(countHelper1-countHelper2+i*2-step)*m+k];
		    	}

		    	deviceD[(countHelper1-countHelper2/2+i-step-1)*m+row]=sumCD1+sumCD2-sumBD;
		    	//printf("in cuda index %d value %lf: \n",(countHelper1-countHelper2/2+i-step-1)*m+row,deviceD[(countHelper1-countHelper2/2+i-step-1)*m+row]);
		    	//printf("gpu:%lf:",newD[(i*multiplier-1)*m+row]);
		    }

		    }
		    
        }
       //sync the thread before go to the next step.
        __syncthreads();

   /*     if(row==0&&column==0)
        {
            for(int i=0;i<9;i++)
            {
              printf("%lf ",oldD[i]);	
            }
          printf("\n");	
        }*/

    }
//***************************begin of unblock version of cyclic reduction*********************************************************************************//
     __global__ void CalculatePArrayKernel(int step,int blockRow, int blockColumn,float* deviceA, float* deviceB, float* deviceC, float* deviceD)
    {
      int bx=blockIdx.x;
      int by=blockIdx.y;
      int tx=threadIdx.x;
      int ty=threadIdx.y;

      int helper11=pow(2.0,(EXPO+1)*1.0);
      int helper22=pow(2.0,(EXPO-step+1)*1.0);
      int helper44=pow(2.0,(EXPO-step+2)*1.0);
      int helper33=pow(2.0,EXPO*1.0)-1;
        //printf("step is running: %d \n",step);

     // if(helper3<pow(2.0,(EXPO-step)*1.0)-1)
        //--step 1 is special case.
     /*  if((tx!=(blockColumn-1))&&(ty!=(blockRow-1)))-----this is very important branch divergence happen here, need
     //to figure out how exactly cuda works!!
        /*****calcualte A******************/
        int helper1=helper11;
        int helper2=helper22;
        int helper4=helper44;
        int flag=0;//special for step1.
        if(step==1)
        {
            helper1=0;
            helper2=0;
            helper4=0;
            flag=1;
        }

        int helper3=ty*blockColumn+tx+1;
        if(helper3<=(pow(2.0,1.0*(EXPO-step))-1.0))
        {
        float ahelperfora1=deviceA[-step+helper1-helper4+2*(helper3)];
        float ahelperfora2=deviceA[-step+helper1-helper4+2*(helper3)-1];
        float bhelperfora1=deviceB[-step+helper1-helper4+2*(helper3)-1];
        deviceA[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*(ahelperfora1)*ahelperfora2/bhelperfora1;

        //*****calculate C******************/
        float chelperforc1=deviceC[-step+helper1-helper4+2*(helper3)];
        float chelperforc2=deviceC[-step+helper1-helper4+2*(helper3)+1];
        float bhelperforc1=deviceB[-step+helper1-helper4+2*(helper3)+1];
        deviceC[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=-1*chelperforc1*chelperforc2/bhelperforc1;

        //calculate B***********************************************//
        float bhelperforb1=deviceB[-step+helper1-helper4+2*(helper3)];
        float bhelperforb2=deviceB[-step+helper1-helper4+2*(helper3)-1];
        float bhelperforb3=deviceB[-step+helper1-helper4+2*(helper3)+1];
        float ahelperforb1=deviceA[-step+helper1-helper4+2*(helper3)];
        float ahelperforb2=deviceA[-step+helper1-helper4+2*(helper3)+1];
        float chelperforb1=deviceC[-step+helper1-helper4+2*(helper3)-1];
        float chelperforb2=deviceC[-step+helper1-helper4+2*(helper3)];
        deviceB[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=bhelperforb1-ahelperforb1/bhelperforb2*chelperforb1-chelperforb2/bhelperforb3*ahelperforb2;

        //calculate D***************************************************//
        float dhelperford1=deviceD[-step+helper1-helper4+2*(helper3)];
        float dhelperford2=deviceD[-step+helper1-helper4+2*(helper3)-1];
        float dhelperford3=deviceD[-step+helper1-helper4+2*(helper3)+1];
        float ahelperford1=deviceA[-step+helper1-helper4+2*(helper3)];
        float bhelperford1=deviceB[-step+helper1-helper4+2*(helper3)-1];
        float bhelperford2=deviceB[-step+helper1-helper4+2*(helper3)+1];
        float chelperford1=deviceC[-step+helper1-helper4+2*(helper3)];
        deviceD[-1-step+helper1-helper2+helper3+flag*(1+helper33)]=dhelperford1-ahelperford1/bhelperford1*dhelperford2-chelperford1/bhelperford2*dhelperford3;
    /*    for(int i=0;i<6;i++)
        {
        	//printf("cudab %lf \n",deviceB[i]);
        	printf("cudab %lf \n",deviceB[i]);
        }

        for(int i=0;i<6;i++)
        {
        	//printf("cudab %lf \n",deviceB[i]);
        	printf("cudad %lf \n",deviceD[i]);
        }*/
    }

        __syncthreads();
    }
    

        __global__ void BackwardKernel(int k,int blockRow, int blockColumn,float* deviceA, float* deviceB, float* deviceC, float* deviceD,float* deviceFinalX,float initialValue)
     {
      int bx1=blockIdx.x;
      int by1=blockIdx.y;
      int tx1=threadIdx.x;
      int ty1=threadIdx.y;
      //printf("inside of kernle %f \n",deviceFinalX[4]);

      int backhelper1=ty1*blockColumn+tx1+1;
      int backhelper2=2*backhelper1-1;//(int((2*backhelper1-1)*pow(2.0,1.0*(k-1))))/(int)(pow(2.0,(k-1)*1.0));
      int backhelper3=(int)pow(2.0,(EXPO+1)*1.0);
      int backhelper4=(int)pow(2.0,(EXPO-k+2)*1.0);


      int h=(int)(pow(2.0,1.0*(k-1)));

      float backhelperd=deviceD[-k+backhelper3-backhelper4+backhelper2];
      float backhelpera=deviceA[-k+backhelper3-backhelper4+backhelper2];
      float backhelperb=deviceB[-k+backhelper3-backhelper4+backhelper2];
      float backhelperc=deviceC[-k+backhelper3-backhelper4+backhelper2];

      int xindex1=backhelper2*pow(2.0,1.0*(k-1))-h;
      int xindex2=backhelper2*pow(2.0,1.0*(k-1))+h;

      //so thread i will be in charge of (2i-1)*2^(k-1) calculation
      //printf("%d ",int((2*backhelper1-1)*pow(2.0,1.0*(k-1))));
      deviceFinalX[(int)(backhelper2*pow(2.0,1.0*(k-1)))]=(backhelperd-backhelpera*deviceFinalX[xindex1]-backhelperc*deviceFinalX[xindex2])*1.0/backhelperb;

      __syncthreads();
     }
      
//***************************end of not block version of cyclic reduction*********************************************************************************//
    
    int main()
    {


      //matrix size will be 63*63 as our setup
      int m=pow(2,EXPO)-1;
      int loopH=pow(2,EXPO-1);
      int conHelp=4*loopH;
         

      //syntax will follow the  routine in the book
      float *B;
      float *C;
      float *D;
      float *X; //X to store the solution
      float *newB;
      float *newC;
 
      int b=1;
      int a=0;
      int maxBlockSize=16;


      //B and C share the same chuck length 
      int chunkLength=m*m;
      float delta=(b-a)*1.0/(m+1.0);
      float deltaSquare=delta*delta;
      int chunkSize=chunkLength*sizeof(float); 
      //need to store all the version of D, it will be 2^k-k-1 block and each block has m value
      int dLength=(pow(2,EXPO+1)-EXPO-2)*m;
      int dSize=dLength*sizeof(float);
     
     


      B=(float*)malloc(chunkSize);
      C=(float*)malloc(chunkSize);
      D=(float*)malloc(dSize);
      //this is to store the final answer
      X=(float*)malloc(chunkSize);
      newB=(float*)malloc(chunkSize);
      newC=(float*)malloc(chunkSize);
      //newD=(float*)malloc(dSize);

          
      //initilize B

      for(int i=0;i<m;i++)
      {
      	for(int j=0;j<m;j++)
      	{
      		B[i*m+j]=0.0;
      		C[i*m+j]=0.0;
      	}
      }
      

      for(int i=0;i<m;i++)
      {
      	B[i*m+i]=-4.0;
      	if(i!=0)
      	{
         B[i*m+i-1]=1.0;
      	}
      	if(i!=m-1)
      	{
      	 B[i*m+i+1]=1.0;;
      	}
      }
  

      //initilize C
      for(int i=0;i<m;i++)
      {
      	C[i*m+i]=1.0;
      } 
    
    //the first 2^k-1 will be the step 0 initial value
      for(int i=0;i<m;i++)
      {
       for(int j=0;j<m;j++)
       {
       	float x=(j+1)*delta;
       	float y=(i+1)*delta;
       	D[i*m+j]=(2*x*x+2*y*y-2*x-2*y)*deltaSquare;
       	//printf("%lf",D[i*m+j]);
       }
        //printf("\n");
      }
      //other value initilized to be 0 at the beginnig
      for(int i=m*m;i<dLength;i++)
      {
      	D[i]=0.0;
      }

      //initilize x
      for(int i=0;i<m;i++)
      {
      	for(int j=0;j<m;j++)
      	{
      		X[i*m+j]=0.0; 
      	}
      }
      //printf("let test this2:\n");

      float *deviceB,*deviceC,*deviceD,*deviceX,*devicenewB,*devicenewC;
      hipMalloc((void**)&deviceB,chunkSize);
      hipMalloc((void**)&deviceC,chunkSize);
      hipMalloc((void**)&deviceD,dSize);
      hipMalloc((void**)&deviceX,chunkSize);
      hipMalloc((void**)&devicenewB,chunkSize);
      hipMalloc((void**)&devicenewC,chunkSize);
      //cudaMalloc((void**)&devicenewD,chunkSize);

      hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);  //store previous value
      hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(deviceD,D,dSize,hipMemcpyHostToDevice);
      hipMemcpy(deviceX,X,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(devicenewB,newB,chunkSize,hipMemcpyHostToDevice);  //store current stored value
      hipMemcpy(devicenewC,newC,chunkSize,hipMemcpyHostToDevice);
     // cudaMemcpy(devicenewD,newD,chunkSize,cudaMemcpyHostToDevice);
        
        //int gridSize=((m+1)/maxBlockSize)*((m+1)/maxBlockSize); //gridSize for this problem will be 16
         	//dim3 dimGrid(1,gridSize)
      	dim3 dimGrid(1,1);  //since the maximum process we are going to use will be 63*63

        int blockRow=maxBlockSize;//pow(2,EXPO/2); //here will be 8 and 8
        int blockColumn=maxBlockSize;//pow(2,EXPO/2); //here will be 8 and 8
        dim3 dimBlock(blockColumn,blockRow);

      for(int step=1;step<EXPO;step++)
      {
      	//so the logic here will be if step is odd, then it use B,C,D as the old value and new value into newB, newC,newD.
      	//if step is even, then use newB,newC,newD as the old value and put the update value into B,C,D.
      
        //here is to calculate the d(2^(k-1))(K-1) in the book
        CalculateTheD<<<dimGrid,dimBlock>>>(step,deviceB,deviceC,deviceD,deviceX,devicenewB,devicenewC);
      }
      hipDeviceSynchronize();
      //the last step here will be 5 so it will write its new value into newB, newC, newD. 
      hipMemcpy(D,deviceD,dSize,hipMemcpyDeviceToHost);
    /*  for (int i=0;i<m;i++)
      {
      	if(i%8==0)
      	{
      		printf("\n");
      	}
      	printf("%lf ",newD[3+i]);
      }*/
      //release some of the memory
      hipFree(deviceB);
      hipFree(deviceC);
      //cudaFree(deviceD);
      hipFree(devicenewB);
      hipFree(devicenewC);
      //cudaFree(devicenewD);
      
      free(B);
      free(C);
      //free(D);
      free(newB);
      free(newC);
      //free(newD);

      /*cudaMemcpy(deviceB,B,chunkSize,cudaMemcpyHostToDevice);
      cudaMemcpy(deviceC,C,chunkSize,cudaMemcpyHostToDevice);*/

      //z will D in the not block version of cyclic reduction, ZA, ZB, ZC will corresponding to A, B and C
      float *Z,*ZA,*ZB,*ZC,*FinalX;
      int finalLengthX=(int)pow(2,EXPO)+1;
      int chunkLengthZ=(pow(2,EXPO)-1)*2+1;
      int zSize=chunkLengthZ*sizeof(float);
      
      Z=(float*)malloc(zSize);
      ZA=(float*)malloc(zSize);
      ZB=(float*)malloc(zSize);
      ZC=(float*)malloc(zSize);
      FinalX=(float*)malloc(finalLengthX*sizeof(float));  //the first and last one should be know by the boundary condition

      float *deviceZ,*deviceZA,*deviceZB, *deviceZC,*deviceFinalX;
	    hipMalloc((void**)&deviceZ,zSize);
	    hipMalloc((void**)&deviceZA,zSize);
	    hipMalloc((void**)&deviceZB,zSize);
	    hipMalloc((void**)&deviceZC,zSize);
	    hipMalloc((void**)&deviceFinalX,finalLengthX*sizeof(float));


      //set up the matrix step 
      for(int j=1;j<=loopH;j++)
      {
      	//for each j, za,zb,zc all going to be different
      	ZA[0]=0;

      	for(int i=1;i<m;i++)
      	{
      		ZA[i]=1.0;
      	}
      	//else will be 0,since it has been seperate to half and half
        for(int i=m;i<chunkLengthZ;i++)
        {
          ZA[i]=0;
        }

        for(int i=0;i<m;i++)
        {
          ZB[i]=-4.0+2*cos((2.0*j-1.0)/(m+1.0)*PI);
          //printf("zb:%f \n",ZB[i]);
        }
        for(int i=m;i<chunkLengthZ;i++)
        {
          ZB[i]=0;
        }

        ZC[m-1]=0;
        for(int i=0;i<m-1;i++)
        {
            ZC[i]=1.0;
        }
        for(int i=m;i<chunkLengthZ;i++)
        {
          ZC[i]=0;
        }

        //if it is the first step z will be from d, otherwise, z will be from the previous solution of x
        if(j==1)
        {
        	for(int i=0;i<m;i++)
        	{
        		/*Z[i]=newD[(loopH-1)*m+i]*(-1.0);
        		printf("this original one being called? %lf \n",Z[i]);*/
        		Z[i]=D[((int)pow(2.0,EXPO+1.0)-3-EXPO)*m+i]*(-1.0);
				printf("z value: %lf \n",Z[i]);
        	}
        	 for(int i=m;i<chunkLengthZ;i++)
		        {
		          Z[i]=0;
		        }
        }
        else
        {
        	for(int i=0;i<m;i++)
        	{
             //to do this will be x
        		Z[i]=FinalX[i+1];
        		//printf("does this ever called? %lf \n",Z[i]);
        	}
        	 for(int i=m;i<chunkLengthZ;i++)
		        {
		          Z[i]=0;
		        }
        }

        //now need to call the cyclic function to find the solution of x

        hipMemcpy(deviceZ,Z,zSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceZA,ZA,zSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceZB,ZB,zSize,hipMemcpyHostToDevice);
        hipMemcpy(deviceZC,ZC,zSize,hipMemcpyHostToDevice);

        for(int j=1;j<EXPO;j++)
        {
        //the lock size should change, the first step it will need 2^(n-j)-1, so first step will be 3 if n=3
        dim3 dimGrid(1,1);
        int blockRow=pow(2,(EXPO-j)/2);
        //printf("blockrow is :%d \n",blockRow);
        int blockColumn=pow(2,EXPO-j-(EXPO-j)/2);
        //printf("blockColumn is :%d \n",blockColumn);
        dim3 dimBlock(blockColumn,blockRow);
          //in each step the processor being used should decrease should be 2^(n-j)-1 in jth step
        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j,blockRow,blockColumn,deviceZA,deviceZB,deviceZC,deviceZ);

        }

          //backward
        //copy the device vector to host
        //cudaMemcpy(ZA,deviceZA,chunkSize,cudaMemcpyDeviceToHost);
       // sleep(20);
        hipDeviceSynchronize(); //cpu will wait until cuda finish the job, this is such important function!
        hipMemcpy(ZB,deviceZB,zSize,hipMemcpyDeviceToHost);
        /*for(int i=0;i<2*m;i++)
        {
        	printf("zbresult:%lf \n",ZB[i]);
        }*/
        //cudaMemcpy(C,deviceC,chunkSize,cudaMemcpyDeviceToHost);
        hipMemcpy(Z,deviceZ,zSize,hipMemcpyDeviceToHost);
        int lastIndex=(int)pow(2,EXPO+1)-EXPO-3;
        float initialValue=Z[lastIndex]/ZB[lastIndex];
        //printf("initial value: %lf \n",initialValue);
        FinalX[0]=0;
        FinalX[(int)pow(2,EXPO-1)]=initialValue;
        //printf("the value in the middle is: %f and this suppose to close to 0.5 when n goes big! \n",FinalX[(int)pow(2,EXPO-1)]);

        hipMemcpy(deviceFinalX,FinalX,finalLengthX*sizeof(float),hipMemcpyHostToDevice);
        for(int k=EXPO-1;k>=1;k--)
        {
          //so the most one will use 2^(n-k) variable will be covered!
        dim3 dimGrid(1,1);
        int blockRow=pow(2,(EXPO-k)/2);
        int blockColumn=pow(2,EXPO-k-(EXPO-k)/2);
        dim3  dimBlock(blockColumn,blockRow);
        
        BackwardKernel<<<dimGrid,dimBlock>>>(k,blockRow,blockColumn,deviceZA,deviceZB,deviceZC,deviceZ,deviceFinalX,initialValue);
        }
         hipDeviceSynchronize();

        hipMemcpy(FinalX,deviceFinalX,finalLengthX*sizeof(float),hipMemcpyDeviceToHost);
      }

      printf("\n final result for x(2^(k-1) block which should have %d values in it:\n",m);
       for (int i=1;i<finalLengthX-1;i++)
      {
       //this will we stored in X the 2^(k-1) the block.
        X[(loopH-1)*m+i-1]=FinalX[i];
      	printf("index: %d, %lf ",(loopH-1)*m+i-1,FinalX[i]);
      }
    
    //now need to do the block wise backsubstitution based on the formula of 5.4.2.17
     for(int step=EXPO-1;step>=1;step--)
     {
      //based on formula 5.4.2.30
     	//ok this is loop trhough the matrix in 5.4.2.17
     	int help1=pow(2,EXPO-step);
     	int localloopH=pow(2,step-1);
     	int thetaHelper=pow(2,step);
     	//inside of each step, you have this much of sybmatrix to solve
     	for(int backStep=1;backStep<=help1;backStep++)
     	{
     		//factorize B(step-1)
     		//first and last one need to be treat specially, C[j-1] will be just identity matrix here

     	
     	   	//************************************************************//
     	   	                 //this is to loop through the factorization
						     for(int j=1;j<=localloopH;j++)
						      {
						      	//for each j, za,zb,zc all going to be different
						      	ZA[0]=0;

						      	for(int i=1;i<m;i++)
						      	{
						      		ZA[i]=1.0;
						      	}
						      	//else will be 0,since it has been seperate to half and half
						        for(int i=m;i<chunkLengthZ;i++)
						        {
						          ZA[i]=0;
						        }

						        for(int i=0;i<m;i++)
						        {
						          ZB[i]=-4.0+2*cos((2.0*backStep-1.0)/(thetaHelper)*PI);
						          //printf("zb:%f \n",ZB[i]);
						        }
						        for(int i=m;i<chunkLengthZ;i++)
						        {
						          ZB[i]=0;
						        }

						        ZC[m-1]=0;
						        for(int i=0;i<m-1;i++)
						        {
						            ZC[i]=1.0;
						        }
						        for(int i=m;i<chunkLengthZ;i++)
						        {
						          ZC[i]=0;
						        }
						        //if it is the first step z will be from d, otherwise, z will be from the previous solution of x
						        if(j==1)
						        {
						        	//the first backsetp and last backstep will be special
						        	if(backStep==1)
						        	{
                                      //teh first d will be d(t-s)(j-1)-c(j-1)x(t)
								        		for(int i=0;i<m;i++)
								        	{
								        		//Z[i]=D[(loopH-1)*m+i]*(-1.0);
								        		//printf("this original one being called? %lf \n",Z[i]);
								        		Z[i]=D[(conHelp-4*help1-step+1)*m+i]-X[(thetaHelper-1)*m+i];
								        		printf("z value: %lf \n",Z[i]);

								        	}
								        	 for(int i=m;i<chunkLengthZ;i++)
										        {
										          Z[i]=0;
										        }
						        	}
						        	else if(backStep==help1)
						        	{
					        				for(int i=0;i<m;i++)
							        	{
							        		//Z[i]=D[(loopH-1)*m+i]*(-1.0);
							        		//printf("this original one being called? %lf \n",Z[i]);
							        		Z[i]=D[(conHelp-2*help1-1-step)*m+i]-X[(conHelp/2-thetaHelper-1)*m+i];

							        	}
							        	 for(int i=m;i<chunkLengthZ;i++)
									        {
									          Z[i]=0;
									        }

						        	}
						        	else //this is at the middle bakcstep
						        	{
						        			for(int i=0;i<m;i++)
							        	{
							        		//Z[i]=D[(loopH-1)*m+i]*(-1.0);
							        		//printf("this original one being called? %lf \n",Z[i]);
							        		Z[i]=D[(2*backStep-1-step+conHelp-2*help1)*m+i]-X[(backStep*thetaHelper-1)*m+i]-X[((backStep-1)*thetaHelper-1)*m+i];
							        	}
							        	 for(int i=m;i<chunkLengthZ;i++)
									        {
									          Z[i]=0;
									        }

						        	}
						        }
						        else
						        {
						        	for(int i=0;i<m;i++)
						        	{
						             //to do this will be x
						        		Z[i]=FinalX[i+1];
						        		//printf("does this ever called? %lf \n",Z[i]);
						        	}
						        	 for(int i=m;i<chunkLengthZ;i++)
								        {
								          Z[i]=0;
								        }
						        }

						        //now need to call the cyclic function to find the solution of x

						        hipMemcpy(deviceZ,Z,zSize,hipMemcpyHostToDevice);
						        hipMemcpy(deviceZA,ZA,zSize,hipMemcpyHostToDevice);
						        hipMemcpy(deviceZB,ZB,zSize,hipMemcpyHostToDevice);
						        hipMemcpy(deviceZC,ZC,zSize,hipMemcpyHostToDevice);

						        for(int j=1;j<EXPO;j++)
						        {
						        //the lock size should change, the first step it will need 2^(n-j)-1, so first step will be 3 if n=3
						        dim3 dimGrid(1,1);
						        int blockRow=pow(2,(EXPO-j)/2);
						        //printf("blockrow is :%d \n",blockRow);
						        int blockColumn=pow(2,EXPO-j-(EXPO-j)/2);
						        //printf("blockColumn is :%d \n",blockColumn);
						        dim3 dimBlock(blockColumn,blockRow);
						          //in each step the processor being used should decrease should be 2^(n-j)-1 in jth step
						        CalculatePArrayKernel<<<dimGrid,dimBlock>>>(j,blockRow,blockColumn,deviceZA,deviceZB,deviceZC,deviceZ);

						        }

						          //backward
						        //copy the device vector to host
						        //cudaMemcpy(ZA,deviceZA,chunkSize,cudaMemcpyDeviceToHost);
						       // sleep(20);
						        hipDeviceSynchronize(); //cpu will wait until cuda finish the job, this is such important function!
						        hipMemcpy(ZB,deviceZB,zSize,hipMemcpyDeviceToHost);
						     /*   for(int i=0;i<2*m;i++)
						        {
						        	printf("zbresult:%lf \n",ZB[i]);
						        }*/
						        //cudaMemcpy(C,deviceC,chunkSize,cudaMemcpyDeviceToHost);
						        hipMemcpy(Z,deviceZ,zSize,hipMemcpyDeviceToHost);
						        int lastIndex=(int)pow(2,EXPO+1)-EXPO-3;
						        float initialValue=Z[lastIndex]/ZB[lastIndex];
						        //printf("initial value: %lf \n",initialValue);
						        FinalX[0]=0;
						        FinalX[(int)pow(2,EXPO-1)]=initialValue;
						        //printf("the value in the middle is: %f and this suppose to close to 0.5 when n goes big! \n",FinalX[(int)pow(2,EXPO-1)]);

						        hipMemcpy(deviceFinalX,FinalX,finalLengthX*sizeof(float),hipMemcpyHostToDevice);
						        for(int k=EXPO-1;k>=1;k--)
						        {
						          //so the most one will use 2^(n-k) variable will be covered!
						        dim3 dimGrid(1,1);
						        int blockRow=pow(2,(EXPO-k)/2);
						        int blockColumn=pow(2,EXPO-k-(EXPO-k)/2);
						        dim3  dimBlock(blockColumn,blockRow);
						        
						        BackwardKernel<<<dimGrid,dimBlock>>>(k,blockRow,blockColumn,deviceZA,deviceZB,deviceZC,deviceZ,deviceFinalX,initialValue);
						        }
						         hipDeviceSynchronize();

						        hipMemcpy(FinalX,deviceFinalX,finalLengthX*sizeof(float),hipMemcpyDeviceToHost);

						      }
                              printf("\n");
						      for(int i=1;i<finalLengthX-1;i++)
						       {
						       	X[((2*backStep-1)*localloopH-1)*m+i-1]=FinalX[i];
                                printf("%lf \n",FinalX[i]);
						       }

     	   	//************************************************************//
     	}

     }

    /* printf("\n");
       for (int i=0;i<m*m;i++)
      {
       //this will we stored in X the 2^(k-1) the block.
      	if(m%10==0)
      	{
      		printf("\n");
      	}
        printf("[%d]:%lf ",i,X[i]);
      }*/
  }





