     #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #define EXPO 2 //so [0,1] will be break into 2^6 intervals 64*64

    __global__ void CalculateTheD(int step,float* deviceB, float* deviceC, float* deviceD, float* deviceX, float* devicenewB, float* devicenewC, float* devicenewD)
    {
       int blocksize=16;
       int gridsize=gridDim.x; //this will always be 1
       // int gridsize2=gridDim.x;
       // printf("gridsize: %d %d\n", gridsize,gridsize2);
       // __device__ bool myGlobalFlag=true;
       
        int bx=blockIdx.x;
        int by=blockIdx.y;
        int tx=threadIdx.x;
        int ty=threadIdx.y;
        
        int blocktotal=blocksize*blocksize;
        int temp1=by*gridsize+bx;
        int temp2=ty*blocksize+tx;

       int ind=temp1*blocktotal+temp2;

      int m=(int)pow(2.0,EXPO*1.0)-1;
      /*int column=threadIdx.x;
      int row=threadIdx.y;*/
      int row=ind/m;
      int column=ind%m;

      int iloopstep=(int)pow(2.0,(EXPO-step)*1.0)-1;
      int h=(int)pow(2.0,(step-1)*1.0);
      int multiplier=(int)pow(2.0,step*1.0);

      float* oldB;
      float* oldC;
      float* oldD;

      float* newB;
      float* newC;
      float* newD;

     //this is to exchange which hold the previous value which hold the current value
      if(step%2==0)
      {
      	oldB=devicenewB;
      	oldC=devicenewC;
      	oldD=devicenewD;

      	newB=deviceB;
      	newC=deviceC;
      	newD=deviceD;
      }
      else
      {
      	oldB=deviceB;
      	oldC=deviceC;
      	oldD=deviceD;
        
        newB=devicenewB;
        newC=devicenewC;
        newD=devicenewD;
      }

      //use the device value as old value and store the updated one in to the new value
      if(ind<m*m) //so only the first 63 threads do work and the other one is hanging there
      {
		    float sumBB=0.0;
		    for(int k=0;k<m;k++)
		    {
		      sumBB=sumBB+oldB[row*m+k]*oldB[k*m+column];
		    }
		    float sumCC=0.0;
		    for(int k=0;k<m;k++)
		    {
		      sumCC=sumCC+oldC[row*m+k]*oldC[k*m+column];
		    }

		    //based on formula (5.4.2.15) on book
		    newB[row*m+column]=2*sumCC-sumBB;
		    newC[row*m+column]=sumCC;

		    //now calculate the new d and it needs to loop through i in each block
		    //look at the third formula on 5.4.2.15 on book
		    if(column==0)
		    {
		    	//for calculate d we just need 63 tthreads but B and C we need 63*63 threads
		    	for(int i=1;i<=iloopstep;i++)
		    {

		    	float sumCD1=0.0;
		    	for(int k=0;k<m;k++)
		    	{
		    		sumCD1=sumCD1+oldC[row*m+k]*oldD[(i*multiplier-h-1)*m+k];
		    	}

		    	float sumCD2=0.0;
		    	for(int k=0;k<m;k++)
		    	{
		    		sumCD2=sumCD2+oldC[row*m+k]*oldD[(i*multiplier+h-1)*m+k];
		    	}

		    	float sumBD=0.0;
		    	for(int k=0;k<m;k++)
		    	{
		           sumBD=sumBD+oldB[row*m+k]*oldD[(i*multiplier-1)*m+k];
		    	}

		    	newD[(i*multiplier-1)*m+row]=sumCD1+sumCD2-sumBD;
		    	//printf("gpu:%lf:",newD[(i*multiplier-1)*m+row]);
		    }

		    }
		    
        }
       //sync the thread before go to the next step.
        __syncthreads();

   /*     if(row==0&&column==0)
        {
            for(int i=0;i<9;i++)
            {
              printf("%lf ",oldD[i]);	
            }
          printf("\n");	
        }*/

    }
       
      

    
    int main()
    {


      //matrix size will be 63*63 as our setup
      int m=pow(2,EXPO)-1;
         

      //syntax will follow the  routine in the book
      float *B;
      float *C;
      float *D;
      float *X; //X to store the solution
      float *newB;
      float *newC;
      float *newD;
      int b=1;
      int a=0;
      int maxBlockSize=16;

      //B and C share the same chuck length 
      int chunkLength=m*m;
      float delta=(b-a)*1.0/(m+1.0);
      float deltaSquare=delta*delta;
      int chunkSize=chunkLength*sizeof(float); 
     // printf("value of m %d and delta %lf!! \n",m,delta);
     


      B=(float*)malloc(chunkSize);
      C=(float*)malloc(chunkSize);
      D=(float*)malloc(chunkSize);
      X=(float*)malloc(chunkSize);
      newB=(float*)malloc(chunkSize);
      newC=(float*)malloc(chunkSize);
      newD=(float*)malloc(chunkSize);

          
      //initilize B

      for(int i=0;i<m;i++)
      {
      	for(int j=0;j<m;j++)
      	{
      		B[i*m+j]=0.0;
      		C[i*m+j]=0.0;
      	}
      }
      

      for(int i=0;i<m;i++)
      {
      	B[i*m+i]=-4.0;
      	if(i!=0)
      	{
         B[i*m+i-1]=1.0;
      	}
      	if(i!=m-1)
      	{
      	 B[i*m+i+1]=1.0;;
      	}
      }
  

      //initilize C
      for(int i=0;i<m;i++)
      {
      	C[i*m+i]=1.0;
      } 
    
     
      for(int i=0;i<m;i++)
      {
       for(int j=0;j<m;j++)
       {
       	float x=(j+1)*delta;
       	float y=(i+1)*delta;
       	D[i*m+j]=(2*x*x+2*y*y-2*x-2*y)*deltaSquare;
       	//printf("%lf",D[i*m+j]);
       }
        //printf("\n");
      }
   
      for(int i=0;i<m;i++)
      {
      	for(int j=0;j<m;j++)
      	{
      		X[i*m+j]=0.0; 
      	}
      }
      //printf("let test this2:\n");

      float *deviceB,*deviceC,*deviceD,*deviceX,*devicenewB,*devicenewC,*devicenewD;
      hipMalloc((void**)&deviceB,chunkSize);
      hipMalloc((void**)&deviceC,chunkSize);
      hipMalloc((void**)&deviceD,chunkSize);
      hipMalloc((void**)&deviceX,chunkSize);
      hipMalloc((void**)&devicenewB,chunkSize);
      hipMalloc((void**)&devicenewC,chunkSize);
      hipMalloc((void**)&devicenewD,chunkSize);

      hipMemcpy(deviceB,B,chunkSize,hipMemcpyHostToDevice);  //store previous value
      hipMemcpy(deviceC,C,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(deviceD,D,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(deviceX,X,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(devicenewB,newB,chunkSize,hipMemcpyHostToDevice);  //store current stored value
      hipMemcpy(devicenewC,newC,chunkSize,hipMemcpyHostToDevice);
      hipMemcpy(devicenewD,newD,chunkSize,hipMemcpyHostToDevice);
        
        //int gridSize=((m+1)/maxBlockSize)*((m+1)/maxBlockSize); //gridSize for this problem will be 16
      	dim3 dimGrid(1,1);  //since the maximum process we are going to use will be 63*63
      	//int blockRow=maxBlockSize;
      	int blockRow=maxBlockSize;//pow(2,EXPO/2); //here will be 8 and 8
        int blockColumn=maxBlockSize;//pow(2,EXPO/2); //here will be 8 and 8
        dim3 dimBlock(blockColumn,blockRow);

      for(int step=1;step<EXPO;step++)
      {
      	//so the logic here will be if step is odd, then it use B,C,D as the old value and new value into newB, newC,newD.
      	//if step is even, then use newB,newC,newD as the old value and put the update value into B,C,D.
      
        //here is to calculate the d(2^(k-1))(K-1) in the book
        CalculateTheD<<<dimGrid,dimBlock>>>(step,deviceB,deviceC,deviceD,deviceX,devicenewB,devicenewC,devicenewD);
      }

      //the last step here will be 5 so it will write its new value into newB, newC, newD. 

      hipMemcpy(newD,devicenewD,chunkSize,hipMemcpyDeviceToHost);
      for (int i=0;i<m;i++)
      {
      	if(i%8==0)
      	{
      		printf("\n");
      	}
        printf("%lf ",newD[m+i]);
      }

      //the value of D we are look at is 

      
    /*  for(int i=0;i<chunkLength;i++)
      {
      	printf("%lf ",newB[i]);
      	if(i==32*63)
      		printf("hello! \n");
      	if(i%63==0)
      	{
      		printf("\n");
      	}
      }*/
      //printf("what about this time %lf:\n",D[63]);

    }


